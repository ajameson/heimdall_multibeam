#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "hd/merge_candidates.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/count.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/discard_iterator.h>

typedef thrust::tuple<hd_float,
                      hd_size,hd_size,hd_size,
                      hd_size,hd_size,hd_size> candidate_tuple;
struct merge_candidates_functor : public thrust::binary_function<candidate_tuple,
                                                                 candidate_tuple,
                                                                 candidate_tuple> {
	inline __host__ __device__
	candidate_tuple operator()(const candidate_tuple& c1,
	                           const candidate_tuple& c2) const {
		hd_float snr1 = thrust::get<0>(c1);
		hd_size  ind1 = thrust::get<1>(c1);
		hd_size  begin1 = thrust::get<2>(c1);
		hd_size  end1 = thrust::get<3>(c1);
		hd_size  filter_ind1 = thrust::get<4>(c1);
		hd_size  dm_ind1 = thrust::get<5>(c1);
		hd_size  members1 = thrust::get<6>(c1);
		
		hd_float snr2 = thrust::get<0>(c2);
		hd_size  ind2 = thrust::get<1>(c2);
		hd_size  begin2 = thrust::get<2>(c2);
		hd_size  end2 = thrust::get<3>(c2);
		hd_size  filter_ind2 = thrust::get<4>(c2);
		hd_size  dm_ind2 = thrust::get<5>(c2);
		hd_size  members2 = thrust::get<6>(c2);
		
		if( snr1 >= snr2 ) {
			return thrust::make_tuple(snr1,
			                          ind1,
			                          //(begin1+begin2)/2,
			                          //(end1+end2)/2,
			                          // TODO: I think this is what gtools does
			                          //min((int)begin1, (int)begin2),
			                          //max((int)end1, (int)end2),
			                          // TODO: But this may be better
			                          begin1,
			                          end1,
			                          filter_ind1,
			                          dm_ind1,
			                          members1+members2);
		}
		else {
			return thrust::make_tuple(snr2,
			                          ind2,
			                          //(begin1+begin2)/2,
			                          //(end1+end2)/2,
			                          //min((int)begin1, (int)begin2),
			                          //max((int)end1, (int)end2),
			                          begin2,
			                          end2,
			                          filter_ind2,
			                          dm_ind2,
			                          members1+members2);
		}
	}
};

hd_error merge_candidates(hd_size            count,
                          hd_size*           d_labels,
                          ConstRawCandidates d_cands,
                          RawCandidates      d_groups)
{
	typedef thrust::device_ptr<hd_float> float_iterator;
	typedef thrust::device_ptr<hd_size>  size_iterator;
	typedef thrust::device_ptr<const hd_float> const_float_iterator;
	typedef thrust::device_ptr<const hd_size>  const_size_iterator;
	
	size_iterator  labels_begin(d_labels);
	
	const_float_iterator cand_peaks_begin(d_cands.peaks);
	const_size_iterator  cand_inds_begin(d_cands.inds);
	const_size_iterator  cand_begins_begin(d_cands.begins);
	const_size_iterator  cand_ends_begin(d_cands.ends);
	const_size_iterator  cand_filter_inds_begin(d_cands.filter_inds);
	const_size_iterator  cand_dm_inds_begin(d_cands.dm_inds);
	const_size_iterator  cand_members_begin(d_cands.members);
	
	float_iterator group_peaks_begin(d_groups.peaks);
	size_iterator  group_inds_begin(d_groups.inds);
	size_iterator  group_begins_begin(d_groups.begins);
	size_iterator  group_ends_begin(d_groups.ends);
	size_iterator  group_filter_inds_begin(d_groups.filter_inds);
	size_iterator  group_dm_inds_begin(d_groups.dm_inds);
	size_iterator  group_members_begin(d_groups.members);
	
	// Sort by labels and remember permutation
	thrust::device_vector<hd_size> d_permutation(count);
	thrust::sequence(d_permutation.begin(), d_permutation.end());
	thrust::sort_by_key(labels_begin, labels_begin + count,
	                    d_permutation.begin());
	
	// Merge giants into groups according to the label
	using thrust::reduce_by_key;
	using thrust::make_zip_iterator;
	using thrust::make_permutation_iterator;
	reduce_by_key(labels_begin, labels_begin + count,
	              make_permutation_iterator(
	                  make_zip_iterator(thrust::make_tuple(cand_peaks_begin,
	                                                       cand_inds_begin,
	                                                       cand_begins_begin,
	                                                       cand_ends_begin,
	                                                       cand_filter_inds_begin,
	                                                       cand_dm_inds_begin,
	                                                       cand_members_begin)),
	              d_permutation.begin()),
	              thrust::make_discard_iterator(), // keys output
	              make_zip_iterator(thrust::make_tuple(group_peaks_begin,
	                                                   group_inds_begin,
	                                                   group_begins_begin,
	                                                   group_ends_begin,
	                                                   group_filter_inds_begin,
	                                                   group_dm_inds_begin,
	                                                   group_members_begin)),
	              thrust::equal_to<hd_size>(),
	              merge_candidates_functor());
	
	return HD_NO_ERROR;
}
