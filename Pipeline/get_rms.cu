#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "hd/get_rms.h"
#include "hd/median_filter.h"
//#include "hd/write_time_series.h"

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>

#include <sstream>

template<typename T>
struct absolute_val : public thrust::unary_function<T,T> {
	inline __host__ __device__
	T operator()(T x) const { return abs(x); }
};

class GetRMSPlan_impl {
	thrust::device_vector<hd_float> buf1;
	thrust::device_vector<hd_float> buf2;
public:

  // This algorithm works by taking the absolute values of the data
  //   and then repeatedly scrunching them using median-of-5 in order
  //   to approximate the median absolute deviation. The RMS is then
  //   just 1.4862 times this.

	hd_float exec(hd_float* d_data, hd_size count) {

		thrust::device_ptr<hd_float> d_data_begin(d_data);
		
		buf1.resize(count);
		buf2.resize(count/5);
		hd_float* buf1_ptr = thrust::raw_pointer_cast(&buf1[0]);
		hd_float* buf2_ptr = thrust::raw_pointer_cast(&buf2[0]);

		thrust::transform(d_data_begin, d_data_begin+count,
		                  buf1.begin(),
		                  absolute_val<hd_float>());
		
		for( hd_size size=count; size>1; size/=5 ) {
			median_scrunch5(buf1_ptr, size, buf2_ptr);
			std::swap(buf1_ptr, buf2_ptr);
		}

		// Note: Result is now at buf1_ptr
		thrust::device_ptr<hd_float> buf1_begin(buf1_ptr);
		hd_float med_abs_dev = buf1_begin[0];
		hd_float rms = med_abs_dev * 1.4862;
		
		return rms;
	}
};

class GetRMSPlanMB_impl {
  thrust::device_vector<hd_float> buf1;
  thrust::device_vector<hd_float> buf2;
public:

  hd_error exec_multibeam (hd_float* d_data, hd_float * d_rms, 
                           hd_size beam_stride, hd_size beam_count, 
                           hd_size nbeams) {

    thrust::device_ptr<hd_float> d_data_begin(d_data);

    // beam_count are uncorrupted samples
    hd_size count = nbeams * beam_count;

    buf1.resize(count);
    buf2.resize(count/5);

    hd_float* buf1_ptr = thrust::raw_pointer_cast(&buf1[0]);
    hd_float* buf2_ptr = thrust::raw_pointer_cast(&buf2[0]);

    reblock_abs_beam (d_data, beam_stride, beam_count, buf1_ptr, nbeams);

    for (hd_size size=beam_count; size>1; size/=5)
    {
      median_scrunch5_beam(buf1_ptr, size, nbeams, buf2_ptr);
      std::swap(buf1_ptr, buf2_ptr);
    }

    // Note: Result is now at buf1_ptr
    thrust::device_ptr<hd_float> buf1_begin(buf1_ptr);
    thrust::device_ptr<hd_float> d_rms_begin(d_rms);

    // Convert to RMS 
    thrust::transform (buf1_begin, buf1_begin + nbeams,
                       thrust::make_constant_iterator(1.4862),
                       d_rms_begin,
                       thrust::multiplies<hd_float>());

    return HD_NO_ERROR;
  }

  // compute single RMS across block, discard overlapping regions between beams
  // useful if original timeseries has already been normalised, for wider filters
  hd_float exec_multibeam (hd_float* d_data,
                           hd_size beam_stride, hd_size beam_count,
                           hd_size nbeams) {

    thrust::device_ptr<hd_float> d_data_begin(d_data);
    hd_size count = nbeams * beam_count;

    buf1.resize(count);
    buf2.resize(count/5);

    hd_float* buf1_ptr = thrust::raw_pointer_cast(&buf1[0]);
    hd_float* buf2_ptr = thrust::raw_pointer_cast(&buf2[0]);

    reblock_abs_beam (d_data, beam_stride, beam_count, buf1_ptr, nbeams);

    for (hd_size size=count; size>1; size/=5)
    {
      median_scrunch5(buf1_ptr, size, buf2_ptr);
      std::swap(buf1_ptr, buf2_ptr);
    }

    // Note: Result is now at buf1_ptr
    thrust::device_ptr<hd_float> buf1_begin(buf1_ptr);
    hd_float med_abs_dev = buf1_begin[0];
    hd_float rms = med_abs_dev * 1.4862;

    return rms;
  }

};

// Public interface (wrapper for implementation)
GetRMSPlan::GetRMSPlan()
	: m_impl(new GetRMSPlan_impl) {}
hd_float GetRMSPlan::exec(hd_float* d_data, hd_size count) {
	return m_impl->exec(d_data, count);
}

GetRMSPlanMB::GetRMSPlanMB()
  : m_impl(new GetRMSPlanMB_impl) {}
hd_error GetRMSPlanMB::exec_multibeam(hd_float* d_data, hd_float* d_rms, hd_size beam_stride, hd_size beam_count, hd_size nbeams) {
  return m_impl->exec_multibeam(d_data, d_rms, beam_stride, beam_count, nbeams);
}
hd_float GetRMSPlanMB::exec_multibeam(hd_float* d_data, hd_size beam_stride, hd_size beam_count, hd_size nbeams) {
  return m_impl->exec_multibeam(d_data, beam_stride, beam_count, nbeams);
}

// Convenience functions for one-off calls
hd_float get_rms(hd_float* d_data, hd_size count) {
	return GetRMSPlan().exec(d_data, count);
}

hd_error get_rms_multibeam (hd_float* d_data, hd_float* d_rms, hd_size beam_stride, hd_size beam_count, hd_size nbeams) {
  return GetRMSPlanMB().exec_multibeam(d_data, d_rms, beam_stride, beam_count, nbeams);
}

hd_error normalise(hd_float* d_data, hd_size count)
{
	thrust::device_ptr<hd_float> d_data_begin(d_data);
	thrust::device_ptr<hd_float> d_data_end(d_data + count);
	
	hd_float rms = get_rms(d_data, count);
	thrust::transform(d_data_begin, d_data_end,
	                  thrust::make_constant_iterator(hd_float(1.0)/rms),
	                  d_data_begin,
	                  thrust::multiplies<hd_float>());
	
	return HD_NO_ERROR;
}

struct normalise_beam_kernel
  : public thrust::unary_function<hd_float,hd_float> {
  const hd_float* in;
  const hd_float* rms;
  const hd_size   size;
  normalise_beam_kernel(const hd_float* in_, const hd_float* rms_, hd_size size_)
    : in(in_), rms(rms_), size(size_) {}
  inline __host__ __device__
  hd_float operator()(unsigned int i) const {
    hd_size beam = i / size;
    return in[i] / rms[beam];
  }
};


hd_error normalise_multibeam (hd_float* d_data, hd_float block_rms, hd_size count)
{
  thrust::device_ptr<hd_float> d_data_begin(d_data);
  thrust::device_ptr<hd_float> d_data_end(d_data + count);

  thrust::transform(d_data_begin, d_data_end,
                    thrust::make_constant_iterator(hd_float(1.0)/block_rms),
                    d_data_begin,
                    thrust::multiplies<hd_float>());

  return HD_NO_ERROR;
}

hd_error normalise_multibeam (hd_float* d_data, hd_float * d_rms, hd_size beam_stride, hd_size cur_nsamps, hd_size nbeams)
{
  thrust::device_ptr<hd_float> d_data_begin(d_data);
  using thrust::make_counting_iterator;
  thrust::transform(make_counting_iterator<unsigned int>(0),
                    make_counting_iterator<unsigned int>(cur_nsamps),
                    d_data_begin,
                    normalise_beam_kernel (d_data, d_rms, beam_stride));

  return HD_NO_ERROR;
}

struct reblock_abs_beam_functor
  : public thrust::unary_function<hd_float,hd_float> {
  const hd_float* in;
  hd_size         length;
  hd_float        delta;
  reblock_abs_beam_functor(const hd_float* in_,
                       hd_size in_length,
                       hd_size in_delta)
    : in(in_), length(in_length), delta(in_delta) {}
  inline __host__ __device__
  hd_float operator()(unsigned int o) const {
    hd_size beam = o / length;
    hd_size i = (beam * delta) + o;
    return fabsf(in[i]);
  }
};

// reblock input into output using an input stride and length
hd_error reblock_abs_beam (const hd_float* d_in,
                           hd_size         in_stride,
                           hd_size         in_length,
                           hd_float *      d_out,
                           hd_size         nbeam)
 {
  using thrust::make_counting_iterator;
  hd_size out_count = in_length * nbeam;
  thrust::device_ptr<hd_float> d_out_begin(d_out);
  hd_size delta = in_stride - in_length;

  thrust::transform(make_counting_iterator<unsigned int>(0),
                    make_counting_iterator<unsigned int>(out_count),
                    d_out_begin,
                    reblock_abs_beam_functor(d_in, in_length, delta));
  return HD_NO_ERROR;
}

