#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "hd/get_rms.h"
#include "hd/median_filter.h"
//#include "hd/write_time_series.h"

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>

#include <sstream>

template<typename T>
struct absolute_val : public thrust::unary_function<T,T> {
	inline __host__ __device__
	T operator()(T x) const { return abs(x); }
};

class GetRMSPlan_impl {
	thrust::device_vector<hd_float> buf1;
	thrust::device_vector<hd_float> buf2;
public:

  // This algorithm works by taking the absolute values of the data
  //   and then repeatedly scrunching them using median-of-5 in order
  //   to approximate the median absolute deviation. The RMS is then
  //   just 1.4862 times this.

	hd_float exec(hd_float* d_data, hd_size count) {

		thrust::device_ptr<hd_float> d_data_begin(d_data);
		
		buf1.resize(count);
		buf2.resize(count/5);
		hd_float* buf1_ptr = thrust::raw_pointer_cast(&buf1[0]);
		hd_float* buf2_ptr = thrust::raw_pointer_cast(&buf2[0]);

		thrust::transform(d_data_begin, d_data_begin+count,
		                  buf1.begin(),
		                  absolute_val<hd_float>());
		
		for( hd_size size=count; size>1; size/=5 ) {
			median_scrunch5(buf1_ptr, size, buf2_ptr);
			std::swap(buf1_ptr, buf2_ptr);
		}

		// Note: Result is now at buf1_ptr
		thrust::device_ptr<hd_float> buf1_begin(buf1_ptr);
		hd_float med_abs_dev = buf1_begin[0];
		hd_float rms = med_abs_dev * 1.4862;
		
		return rms;
	}
};

class GetRMSPlanMB_impl {
  thrust::device_vector<hd_float> buf1;
  thrust::device_vector<hd_float> buf2;
public:

  hd_error exec_multibeam (hd_float* d_data, hd_float * d_rms, 
                           hd_size beam_stride, hd_size beam_count, 
                           hd_size nbeams) {

    thrust::device_ptr<hd_float> d_data_begin(d_data);

    // includes corrupted overlap regions
    //hd_size dm_delay    = beam_stride - beam_count;
    //hd_size count_dirty = nbeams * beam_stride - dm_delay;
    hd_size count_clean = nbeams * beam_count;

    buf1.resize(count_clean);
    buf2.resize(count_clean/5);

    hd_float* buf1_ptr = thrust::raw_pointer_cast(&buf1[0]);
    hd_float* buf2_ptr = thrust::raw_pointer_cast(&buf2[0]);

    // slower
    for (unsigned ibeam=0; ibeam<nbeams; ibeam++)
    {
      hd_size stride_offset = ibeam * beam_stride;
      hd_size count_offset = ibeam * beam_count;

      thrust::transform(d_data_begin + stride_offset, 
                        d_data_begin + stride_offset + beam_count,
                        buf1.begin() + count_offset,
                        absolute_val<hd_float>());
    }

    for (hd_size size=beam_count; size>1; size/=5)
    {
      median_scrunch5_beam(buf1_ptr, size, nbeams, buf2_ptr);
      std::swap(buf1_ptr, buf2_ptr);
    }

    // Note: Result is now at buf1_ptr
    thrust::device_ptr<hd_float> buf1_begin(buf1_ptr);
    thrust::device_ptr<hd_float> d_rms_begin(d_rms);

    // Convert to RMS 
    thrust::transform (buf1_begin, buf1_begin + nbeams,
                       thrust::make_constant_iterator(1.4862),
                       d_rms_begin,
                       thrust::multiplies<hd_float>());

    return HD_NO_ERROR;
  }

  // compute single RMS across block, discard overlapping regions between beams
  // useful if original timeseries has already been normalised, for wider filters
  hd_float exec_multibeam (hd_float* d_data,
                           hd_size beam_stride, hd_size beam_count,
                           hd_size nbeams) {

    thrust::device_ptr<hd_float> d_data_begin(d_data);

    hd_size count = nbeams * beam_count;

    buf1.resize(count);
    buf2.resize(count/5);

    hd_float* buf1_ptr = thrust::raw_pointer_cast(&buf1[0]);
    hd_float* buf2_ptr = thrust::raw_pointer_cast(&buf2[0]);

    for (unsigned ibeam=0; ibeam<nbeams; ibeam++)
    {
      hd_size stride_offset = ibeam * beam_stride;
      hd_size count_offset = ibeam * beam_count;

      thrust::transform(d_data_begin + stride_offset,
                        d_data_begin + stride_offset + beam_count,
                        buf1.begin() + count_offset,
                        absolute_val<hd_float>());
    }

    for (hd_size size=count; size>1; size/=5)
    {
      median_scrunch5(buf1_ptr, size, buf2_ptr);
      std::swap(buf1_ptr, buf2_ptr);
    }

    // Note: Result is now at buf1_ptr
    thrust::device_ptr<hd_float> buf1_begin(buf1_ptr);
    hd_float med_abs_dev = buf1_begin[0];
    hd_float rms = med_abs_dev * 1.4862;

    return rms;
  }

};

// Public interface (wrapper for implementation)
GetRMSPlan::GetRMSPlan()
	: m_impl(new GetRMSPlan_impl) {}
hd_float GetRMSPlan::exec(hd_float* d_data, hd_size count) {
	return m_impl->exec(d_data, count);
}

GetRMSPlanMB::GetRMSPlanMB()
  : m_impl(new GetRMSPlanMB_impl) {}
hd_error GetRMSPlanMB::exec_multibeam(hd_float* d_data, hd_float* d_rms, hd_size beam_stride, hd_size beam_count, hd_size nbeams) {
  return m_impl->exec_multibeam(d_data, d_rms, beam_stride, beam_count, nbeams);
}
hd_float GetRMSPlanMB::exec_multibeam(hd_float* d_data, hd_size beam_stride, hd_size beam_count, hd_size nbeams) {
  return m_impl->exec_multibeam(d_data, beam_stride, beam_count, nbeams);
}

// Convenience functions for one-off calls
hd_float get_rms(hd_float* d_data, hd_size count) {
	return GetRMSPlan().exec(d_data, count);
}

hd_error get_rms_multibeam (hd_float* d_data, hd_float* d_rms, hd_size beam_stride, hd_size beam_count, hd_size nbeams) {
  return GetRMSPlanMB().exec_multibeam(d_data, d_rms, beam_stride, beam_count, nbeams);
}

hd_error normalise(hd_float* d_data, hd_size count)
{
	thrust::device_ptr<hd_float> d_data_begin(d_data);
	thrust::device_ptr<hd_float> d_data_end(d_data + count);
	
	hd_float rms = get_rms(d_data, count);
	thrust::transform(d_data_begin, d_data_end,
	                  thrust::make_constant_iterator(hd_float(1.0)/rms),
	                  d_data_begin,
	                  thrust::multiplies<hd_float>());
	
	return HD_NO_ERROR;
}

struct normalise_beam_kernel
  : public thrust::unary_function<hd_float,hd_float> {
  const hd_float* in;
  const hd_float* rms;
  const hd_size   size;
  normalise_beam_kernel(const hd_float* in_, const hd_float* rms_, hd_size size_)
    : in(in_), rms(rms_), size(size_) {}
  inline __host__ __device__
  hd_float operator()(unsigned int i) const {
    hd_size beam = i / size;
    return in[i] / rms[beam];
  }
};


hd_error normalise_multibeam (hd_float* d_data, hd_float block_rms, hd_size count)
{
  thrust::device_ptr<hd_float> d_data_begin(d_data);
  thrust::device_ptr<hd_float> d_data_end(d_data + count);

  thrust::transform(d_data_begin, d_data_end,
                    thrust::make_constant_iterator(hd_float(1.0)/block_rms),
                    d_data_begin,
                    thrust::multiplies<hd_float>());

  return HD_NO_ERROR;
}

hd_error normalise_multibeam (hd_float* d_data, hd_float * d_rms, hd_size beam_stride, hd_size cur_nsamps, hd_size nbeams)
{
  thrust::device_ptr<hd_float> d_data_begin(d_data);
  using thrust::make_counting_iterator;
  thrust::transform(make_counting_iterator<unsigned int>(0),
                    make_counting_iterator<unsigned int>(cur_nsamps),
                    d_data_begin,
                    normalise_beam_kernel (d_data, d_rms, beam_stride));

  return HD_NO_ERROR;
}

