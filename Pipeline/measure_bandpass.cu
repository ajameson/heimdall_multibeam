#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "hd/measure_bandpass.h"
#include "hd/median_filter.h"
#include "hd/get_rms.h"
#include <stdio.h>

// TESTING ONLY
// #include "hd/write_time_series.h"

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/random.h>

template<typename WordType>
struct unpack_functor : public thrust::unary_function<unsigned int, float> {
	const WordType* in;
	unsigned int    nbits;
	unsigned int    chans_per_word;
	WordType        bitmask;
	unpack_functor(const WordType* in_, unsigned int nbits_)
		: in(in_), nbits(nbits_),
		  chans_per_word(sizeof(WordType)*8/nbits), bitmask((1<<nbits)-1) {}
	
	inline __host__ __device__
	float operator()(unsigned int c) const {
		unsigned int w = c / chans_per_word;
		unsigned int k = c % chans_per_word;
		return (float)((in[w] >> (k*nbits)) & bitmask);
	}
};

template<typename T>
struct abs_val : public thrust::unary_function<T,T> {
	inline __host__ __device__
	T operator()(T x) const { return fabs(x); }
};
hd_error measure_bandpass(const hd_byte* d_filterbank,
                          hd_size        nsamps,
                          hd_size        nchans,
                          hd_size        nbits,
                          hd_float*      d_bandpass,
                          hd_float*      rms)
{

	using thrust::make_counting_iterator;
	
	typedef unsigned int WordType;
	hd_size stride = nchans * nbits/8 / sizeof(WordType);
	//stride=7;
	//printf("VR says %d %d %d\n",stride,nsamps,sizeof(WordType));	

	//thrust::device_vector<hd_float> d_spectrum(nchans);
	//hd_float* d_spectrum_ptr = thrust::raw_pointer_cast(&d_spectrum[0]);
	thrust::device_ptr<hd_float> d_bandpass_begin(d_bandpass);
	
  //std::cerr << "measure_bandpass 1" << std::endl;

	// First we find the median of a selection of sample spectra
	// TODO: Can/should make this a parameter?
	// TODO: Does this give a good balance of performance vs. accuracy?
	// Note: Changing this requires changing the code below.
	hd_size spectrum_count = 5*5*5 *5*5;
	
	thrust::device_vector<hd_float> d_sample_spectra1(spectrum_count*nchans);
	thrust::device_vector<hd_float> d_sample_spectra2(spectrum_count/5*nchans);
	thrust::device_vector<hd_float> d_sample_spectra3(spectrum_count/5/5*nchans);
	thrust::device_vector<hd_float> d_sample_spectra4(spectrum_count/5/5/5*nchans);
	thrust::device_vector<hd_float> d_sample_spectra5(spectrum_count/5/5/5/5*nchans);
						
	hd_float* d_sample_spectra1_ptr =
		thrust::raw_pointer_cast(&d_sample_spectra1[0]);
	hd_float* d_sample_spectra2_ptr =
		thrust::raw_pointer_cast(&d_sample_spectra2[0]);
	hd_float* d_sample_spectra3_ptr =
		thrust::raw_pointer_cast(&d_sample_spectra3[0]);
	hd_float* d_sample_spectra4_ptr =
		thrust::raw_pointer_cast(&d_sample_spectra4[0]);
	hd_float* d_sample_spectra5_ptr =
		thrust::raw_pointer_cast(&d_sample_spectra5[0]);

	// TODO: Make this more random?
	hd_size seed = 123456;
	thrust::default_random_engine rng(seed);
	thrust::uniform_int_distribution<unsigned int> distribution(0, nsamps-1);
	// Extract spectrum_count sample spectra from the filterbank

	for( hd_size i=0; i<spectrum_count; ++i ) {
		//hd_size t = i * spectrum_stride; // Regular spacing		
		hd_size t = distribution(rng); // Uniform random sampling
		/* IN BELOW, NCHANS WAS STRIDE */
		WordType* d_in = (WordType*)&d_filterbank[t*nchans];	 
		thrust::transform(make_counting_iterator<unsigned int>(0),
		                  make_counting_iterator<unsigned int>(nchans),
		                  d_sample_spectra1.begin() + i*nchans,
		                  unpack_functor<WordType>(d_in, nbits));
	}


	
  //std::cerr << "measure_bandpass 1.1 nchans=" << nchans << " spectrum_count=" << spectrum_count << " ptr1=" << (void *) d_sample_spectra1_ptr << " ptr2=" << (void *) d_sample_spectra2_ptr << std::endl;

	// Compute the 'remedian' (recursive median) of the sample spectra
	// Note: We do this instead of a proper median for performance and simplicity
	//std::cerr << "got here VR" << std::endl;
	median_scrunch5_array(d_sample_spectra1_ptr, nchans,
	                      spectrum_count,
	                      d_sample_spectra2_ptr);
  //std::cerr << "measure_bandpass 1.1.1" << std::endl;
	median_scrunch5_array(d_sample_spectra2_ptr, nchans,
	                      spectrum_count / 5,
	                      d_sample_spectra3_ptr);
  //std::cerr << "measure_bandpass 1.1.1" << std::endl;
	median_scrunch5_array(d_sample_spectra3_ptr, nchans,
	                      spectrum_count / 5 / 5,
	                      d_sample_spectra4_ptr);
  //std::cerr << "measure_bandpass 1.1.1" << std::endl;
	median_scrunch5_array(d_sample_spectra4_ptr, nchans,
	                      spectrum_count / 5 / 5 / 5,
	                      d_sample_spectra5_ptr);
  //std::cerr << "measure_bandpass 1.1.1" << std::endl;
	median_scrunch5_array(d_sample_spectra5_ptr, nchans,
	                      spectrum_count / 5 / 5 / 5 / 5,
	                      d_bandpass);
	

  //std::cerr << "measure_bandpass 1.2" << std::endl;

	//write_device_time_series(d_bandpass, nchans, 1.f, "median_spectrum.tim");
	
	// Now we smooth the spectrum to produce an estimate of the bandpass
	thrust::device_vector<hd_float> d_scrunched_spectrum(nchans/5);
	hd_float* d_scrunched_spectrum_ptr =
		thrust::raw_pointer_cast(&d_scrunched_spectrum[0]);
	// TODO: This algorithm was derived empirically. It may not be suitable
	//         if applied to a different observing setup.
	median_scrunch5(d_bandpass, nchans,
	                d_scrunched_spectrum_ptr);
	median_filter5(d_scrunched_spectrum_ptr, nchans / 5,
	               d_bandpass);
	mean_filter2(d_bandpass, nchans / 5,
	             d_scrunched_spectrum_ptr);
	linear_stretch(d_scrunched_spectrum_ptr, nchans / 5,
	               d_bandpass,
	               // Note: We must use the truncate-rounded length
	               nchans / 5 * 5);
	
	// Extrapolate to make up the truncated samples
	// Note: This is very inefficient, but shouldn't affect performance
	for( hd_size i=nchans/5*5; i<nchans; ++i ) {
		d_bandpass_begin[i] =
			2 * d_bandpass_begin[i-1] - d_bandpass_begin[i-2];
	}
	// The bandpass estimate is now in d_bandpass
	
	//write_device_time_series(d_bandpass, nchans, 1.f, "bandpass.tim");
	
	// Now we estimate the RMS in the bandpass
	// ---------------------------------------
	//std::vector<hd_float> sample_rms1(spectrum_count);
	//std::vector<hd_float> sample_rms2(spectrum_count);
	// TODO: These are on the device only because I couldn't be bothered
	//         making host versions of the median_scrunch functions.
	thrust::device_vector<hd_float> d_sample_rms1(spectrum_count);
	thrust::device_vector<hd_float> d_sample_rms2(spectrum_count);
	hd_float* d_sample_rms1_ptr =
		thrust::raw_pointer_cast(&d_sample_rms1[0]);
	hd_float* d_sample_rms2_ptr =
		thrust::raw_pointer_cast(&d_sample_rms2[0]);
	
	for( hd_size i=0; i<spectrum_count; ++i ) {
		// Subtract the bandpass from the spectrum
		thrust::transform(d_sample_spectra1.begin() + i*nchans,
		                  d_sample_spectra1.begin() + (i+1)*nchans,
		                  d_bandpass_begin,
		                  d_sample_spectra1.begin() + i*nchans,
		                  thrust::minus<hd_float>());
		// Take the absolute value
		thrust::transform(d_sample_spectra1.begin() + i*nchans,
		                  d_sample_spectra1.begin() + (i+1)*nchans,
		                  d_sample_spectra1.begin() + i*nchans,
		                  abs_val<hd_float>());
		
		//d_sample_rms1[i] = get_rms(d_sample_spectra1_ptr + i*nchans, nchans);
	}
	
  //std::cerr << "measure_bandpass 2" << std::endl;

	thrust::device_vector<hd_float> d_mad(nchans);
	hd_float* d_mad_ptr = thrust::raw_pointer_cast(&d_mad[0]);
	
	// Compute the 'remedian' (recursive median) of the sample spectra
	// Note: We do this instead of a proper median for performance and simplicity
	median_scrunch5_array(d_sample_spectra1_ptr, nchans,
	                      spectrum_count,
	                      d_sample_spectra2_ptr);
	median_scrunch5_array(d_sample_spectra2_ptr, nchans,
	                      spectrum_count / 5,
	                      d_sample_spectra3_ptr);
	median_scrunch5_array(d_sample_spectra3_ptr, nchans,
	                      spectrum_count / 5 / 5,
	                      d_sample_spectra4_ptr);
	median_scrunch5_array(d_sample_spectra4_ptr, nchans,
	                      spectrum_count / 5 / 5 / 5,
	                      d_sample_spectra5_ptr);
	median_scrunch5_array(d_sample_spectra5_ptr, nchans,
	                      spectrum_count / 5 / 5 / 5 / 5,
	                      d_mad_ptr);
	
	// Convert median absolute deviation to standard deviation
	using namespace thrust::placeholders;
	thrust::transform(d_mad.begin(), d_mad.end(),
	                  d_mad.begin(),
	                  _1 * 1.4826f);
	
	//write_device_time_series(d_mad_ptr, nchans, 1.f, "mad.tim");
	
	// Smooth the band RMS
	median_scrunch5(d_mad_ptr, nchans,
	                d_scrunched_spectrum_ptr);
	median_filter5(d_scrunched_spectrum_ptr, nchans / 5,
	               d_mad_ptr);
	mean_filter2(d_mad_ptr, nchans / 5,
	             d_scrunched_spectrum_ptr);
	linear_stretch(d_scrunched_spectrum_ptr, nchans / 5,
	               d_mad_ptr,
	               // Note: We must use the truncate-rounded length
	               nchans / 5 * 5);
	
	// Extrapolate to make up the truncated samples
	// Note: This is very inefficient, but shouldn't affect performance
	for( hd_size i=nchans/5*5; i<nchans; ++i ) {
		d_mad[i] = 2 * d_mad[i-1] - d_mad[i-2];
	}
	
	// TODO: Do we need to apply narrow-band filtering to (all of the)
	//         time-scrunched versions of the filterbank too?
	//         This would allow us to catch narrow, extended RFI.
	//       What about scrunching in frequency a bit too?
	//         Probably a bad idea, as that's what the broad-band mitigation
	//           is for, and we want as much distinction as possible.
	
	//write_device_time_series(d_mad_ptr, nchans, 1.f, "smooth_mad.tim");
	
  //std::cerr << "measure_bandpass 3" << std::endl;

	// Find the median RMS across the band
	std::vector<hd_float> h_mad(nchans);
	thrust::copy(d_mad.begin(), d_mad.end(), h_mad.begin());
	std::nth_element(h_mad.begin(), h_mad.begin()+h_mad.size()/2, h_mad.end());
	*rms = h_mad[h_mad.size()/2];
	
  //std::cerr << "measure_bandpass 4" << std::endl;

	/*
	// And finally use the remedian to estimate the global RMS
	median_scrunch5(d_sample_rms1_ptr, spectrum_count,
	                d_sample_rms2_ptr);
	median_scrunch5(d_sample_rms2_ptr, spectrum_count / 5,
	                d_sample_rms1_ptr);
	median_scrunch5(d_sample_rms1_ptr, spectrum_count / 5 / 5,
	d_sample_rms2_ptr);
	*rms = d_sample_rms2[0];
	*/
	// ---------------------------------------
	
	return HD_NO_ERROR;
}

/*
// TODO: The below code is work in progress.

hd_error measure_band_avg(const hd_byte* d_filterbank,
                          hd_size        nsamps,
                          hd_size        nchans,
                          hd_size        nbits,
                          hd_float*      d_band_avg)
{
	using thrust::make_counting_iterator;
	
	typedef unsigned int WordType;
	hd_size stride = nchans * nbits/8 / sizeof(WordType);
	
	thrust::device_vector<hd_float> d_bandpass(nchans);
	hd_float* d_bandpass_ptr = raw_pointer_cast(&d_bandpass[0]);
	hd_float rms;
	measure_bandpass(d_filterbank, nsamps, nchans, nbits,
	                 d_bandpass_ptr, &rms);
	
	// TODO: Check that this gives good results, good performance and not
	//         too much memory use.
	hd_size spectrum_count = 1<<11;
	thrust::device_vector<hd_float> d_sample_spectra(spectrum_count*nchans);
	thrust::device_vector<hd_float> d_scrunched_spectra(spectrum_count/2*nchans);
	using thrust::raw_pointer_cast;
	hd_float* d_sample_spectra_ptr =
		raw_pointer_cast(&d_sample_spectra[0]);
	hd_float* d_scrunched_spectra_ptr =
		raw_pointer_cast(&d_scrunched_spectra[0]);
	// TODO: Make this more random?
	hd_size seed = 123456;
	thrust::default_random_engine rng(seed);
	thrust::uniform_int_distribution<unsigned int> distribution(0, nsamps-1);
	// Extract spectrum_count sample spectra from the filterbank
	for( hd_size i=0; i<spectrum_count; ++i ) {
		//hd_size t = i * spectrum_stride; // Regular spacing
		hd_size t = distribution(rng); // Uniform random sampling
		WordType* d_in = (WordType*)&d_filterbank[t*stride];
		// Extract the spectrum
		thrust::transform(make_counting_iterator<unsigned int>(0),
		                  make_counting_iterator<unsigned int>(nchans),
		                  d_sample_spectra.begin() + i*nchans,
		                  unpack_functor<WordType>(d_in, nbits));
		// Subtract the bandpass
		thrust::transform(d_sample_spectra.begin() + i*nchans,
		                  d_sample_spectra.begin() + (i+1)*nchans,
		                  d_bandpass.begin(),
		                  d_sample_spectra.begin() + i*nchans,
		                  thrust::minus<hd_float>());
	}
	
	for( hd_size count=spectrum_count; count>1; count/=2 ) {
		mean_scrunch2_array(d_sample_spectra_ptr, nchans,
		                    count,
		                    d_scrunched_spectra_ptr);
		
		std::swap(d_sample_spectra_ptr, d_scrunched_spectra_ptr);
	}
	thrust::device_ptr<hd_float> d_band_sum_begin(d_sample_spectra_ptr);
	thrust::device_ptr<hd_float> d_band_sum_end = d_band_sum_begin + nchans;
	thrust::device_ptr<hd_float> d_band_avg_begin(d_band_avg);
	using namespace thrust::placeholders;
	thrust::transform(d_band_sum_begin, d_band_sum_end,
	                  thrust::make_constant_iterator((float)sqrt(spectrum_count)),
	                  d_band_avg_begin,
	                  thrust::multiplies<hd_float>());
	                  //_1 * sqrt((float)spectrum_count));
	
	return HD_NO_ERROR;
}
*/
/*
// Measures each channel's outlier fraction, i.e., the fraction of samples
//   in the channel that exceed thresh*rms.
hd_error measure_band_outliers(const hd_byte* d_filterbank,
                               hd_size        nsamps,
                               hd_size        nchans,
                               hd_size        nbits,
                               hd_float       thresh,
                               hd_float*      d_band_outlier_fracs)
{
	using thrust::make_counting_iterator;
	
	typedef unsigned int WordType;
	hd_size stride = nchans * nbits/8 / sizeof(WordType);
	
	thrust::device_vector<hd_float> d_bandpass(nchans);
	hd_float* d_bandpass_ptr = raw_pointer_cast(&d_bandpass[0]);
	hd_float rms;
	measure_bandpass(d_filterbank, nsamps, nchans, nbits,
	                 d_bandpass_ptr, &rms);
	
	// TODO: Check that this gives good results, good performance and not
	//         too much memory use.
	hd_size spectrum_count = 1<<10;
	thrust::device_vector<hd_float> d_sample_spectra(spectrum_count*nchans);
	thrust::device_vector<hd_float> d_scrunched_spectra(spectrum_count/2*nchans);
	using thrust::raw_pointer_cast;
	hd_float* d_sample_spectra_ptr =
		raw_pointer_cast(&d_sample_spectra[0]);
	hd_float* d_scrunched_spectra_ptr =
		raw_pointer_cast(&d_scrunched_spectra[0]);
	// TODO: Make this more random?
	hd_size seed = 123456;
	thrust::default_random_engine rng(seed);
	thrust::uniform_int_distribution<unsigned int> distribution(0, nsamps-1);
	// Extract spectrum_count sample spectra from the filterbank
	for( hd_size i=0; i<spectrum_count; ++i ) {
		//hd_size t = i * spectrum_stride; // Regular spacing
		hd_size t = distribution(rng); // Uniform random sampling
		WordType* d_in = (WordType*)&d_filterbank[t*stride];
		// Extract spectrum
		thrust::transform(make_counting_iterator<unsigned int>(0),
		                  make_counting_iterator<unsigned int>(nchans),
		                  d_sample_spectra.begin() + i*nchans,
		                  unpack_functor<WordType>(d_in, nbits));
		// Transform to a mask of outliers
		thrust::transform(d_sample_spectra.begin() + i*nchans,
		                  d_sample_spectra.begin() + (i+1)*nchans,
		                  d_bandpass.begin(),
		                  d_sample_spectra.begin() + i*nchans,
		                  abs_diff_exceeds<hd_float>(thresh*rms));
		// TODO: transform to mask of fabs(x-xb) > thresh, where xb is the bandpass
		//       Then leave the below scrunching as is, remove the final
		//         normalisation transform, and check if the result exceeds
		//         the desired time infection percentage.
	}
	
	for( hd_size count=spectrum_count; count>1; count/=2 ) {
		mean_scrunch2_array(d_sample_spectra_ptr, nchans,
		                    count,
		                    d_scrunched_spectra_ptr);
		
		std::swap(d_sample_spectra_ptr, d_scrunched_spectra_ptr);
	}
	thrust::device_ptr<hd_float> d_band_sum_begin(d_sample_spectra_ptr);
	thrust::device_ptr<hd_float> d_band_sum_end = d_band_sum_begin + nchans;
	thrust::device_ptr<hd_float> d_band_outlier_fracs_begin(d_band_outlier_fracs);
	using namespace thrust::placeholders;
	
	// Copy to output
	thrust::transform(d_band_sum_begin, d_band_sum_end,
	                  d_band_outlier_fracs_begin,
	                  thrust::identity<hd_float>());
	
	return HD_NO_ERROR;
}
*/
