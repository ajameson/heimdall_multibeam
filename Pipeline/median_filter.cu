/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "hd/median_filter.h"
#include <stdio.h>

#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/counting_iterator.h>

/*
  Note: The implementations of median3-5 here can be derived from
          'sorting networks'.
 */

inline __host__ __device__
float median3(float a, float b, float c) {
	return a < b ? b < c ? b
	                      : a < c ? c : a
	             : a < c ? a
	                     : b < c ? c : b;
}
inline __host__ __device__
float median4(float a, float b, float c, float d) {
	return a < c ? b < d ? a < b ? c < d ? 0.5f*(b+c) : 0.5f*(b+d)
	                             : c < d ? 0.5f*(a+c) : 0.5f*(a+d)
	                     : a < d ? c < b ? 0.5f*(d+c) : 0.5f*(b+d)
	                             : c < b ? 0.5f*(a+c) : 0.5f*(a+b)
	             : b < d ? c < b ? a < d ? 0.5f*(b+a) : 0.5f*(b+d)
	                             : a < d ? 0.5f*(a+c) : 0.5f*(c+d)
	                     : c < d ? a < b ? 0.5f*(d+a) : 0.5f*(b+d)
	                             : a < b ? 0.5f*(a+c) : 0.5f*(c+b);
}
inline __host__ __device__
float median5(float a, float b, float c, float d, float e) {
	// Note: This wicked code is by 'DRBlaise' and was found here:
	//         http://stackoverflow.com/a/2117018
	return b < a ? d < c ? b < d ? a < e ? a < d ? e < d ? e : d
                                                 : c < a ? c : a
                                         : e < d ? a < d ? a : d
                                                 : c < e ? c : e
                                 : c < e ? b < c ? a < c ? a : c
                                                 : e < b ? e : b
                                         : b < e ? a < e ? a : e
                                                 : c < b ? c : b
                         : b < c ? a < e ? a < c ? e < c ? e : c
                                                 : d < a ? d : a
                                         : e < c ? a < c ? a : c
                                                 : d < e ? d : e
                                 : d < e ? b < d ? a < d ? a : d
                                                 : e < b ? e : b
                                         : b < e ? a < e ? a : e
                                                 : d < b ? d : b
                 : d < c ? a < d ? b < e ? b < d ? e < d ? e : d
                                                 : c < b ? c : b
                                         : e < d ? b < d ? b : d
                                                 : c < e ? c : e
                                 : c < e ? a < c ? b < c ? b : c
                                                 : e < a ? e : a
                                         : a < e ? b < e ? b : e
                                                 : c < a ? c : a
                         : a < c ? b < e ? b < c ? e < c ? e : c
                                                 : d < b ? d : b
                                         : e < c ? b < c ? b : c
                                                 : d < e ? d : e
                                 : d < e ? a < d ? b < d ? b : d
                                                 : e < a ? e : a
                                         : a < e ? b < e ? b : e
                                                 : d < a ? d : a;
}

struct median_filter3_kernel
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	unsigned int    count;
	median_filter3_kernel(const hd_float* in_,
	                      unsigned int count_)
		: in(in_), count(count_) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		// Note: We shrink the window near boundaries
		if( i > 0 && i < count-1 ) {
			return median3(in[i-1], in[i], in[i+1]);
		}
		else if( i == 0 ) {
			return 0.5f*(in[i]+in[i+1]);
		}
		else { //if( i == count-1 ) {
			return 0.5f*(in[i]+in[i-1]);
		}
	}
};

struct median_filter5_kernel
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	unsigned int    count;
	median_filter5_kernel(const hd_float* in_,
	                      unsigned int count_)
		: in(in_), count(count_) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		// Note: We shrink the window near boundaries
		if( i > 1 && i < count-2 ) {
			return median5(in[i-2], in[i-1], in[i], in[i+1], in[i+2]);
		}
		else if( i == 0 ) {
			return median3(in[i], in[i+1], in[i+2]);
		}
		else if( i == 1 ) {
			return median4(in[i-1], in[i], in[i+1], in[i+2]);
		}
		else if( i == count-1 ) {
			return median3(in[i], in[i-1], in[i-2]);
		}
		else { //if ( i == count-2 ) {
			return median4(in[i+1], in[i], in[i-1], in[i-2]);
		}
	}
};

struct median_scrunch3_kernel
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	median_scrunch3_kernel(const hd_float* in_)
		: in(in_) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		hd_float a = in[3*i+0];
		hd_float b = in[3*i+1];
		hd_float c = in[3*i+2];
		return median3(a, b, c);
	}
};

struct median_scrunch5_kernel
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	median_scrunch5_kernel(const hd_float* in_)
		: in(in_) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		hd_float a = in[5*i+0];
		hd_float b = in[5*i+1];
		hd_float c = in[5*i+2];
		hd_float d = in[5*i+3];
		hd_float e = in[5*i+4];
		return median5(a, b, c, d, e);
	}
};

struct median_scrunch3_array_kernel
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	const hd_size   size;
	median_scrunch3_array_kernel(const hd_float* in_, hd_size size_)
		: in(in_), size(size_) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		hd_size array = i / size;
		hd_size j     = i % size;
		
		hd_float a = in[(3*array+0)*size + j];
		hd_float b = in[(3*array+1)*size + j];
		hd_float c = in[(3*array+2)*size + j];
		return median3(a, b, c);
	}
};

struct median_scrunch5_array_kernel
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	const hd_size   size;
	median_scrunch5_array_kernel(const hd_float* in_, hd_size size_)
		: in(in_), size(size_) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		hd_size array = i / size;
		hd_size j     = i % size;
		hd_float a = in[(5*array+0)*size + j];
		hd_float b = in[(5*array+1)*size + j];
		hd_float c = in[(5*array+2)*size + j];
		hd_float d = in[(5*array+3)*size + j];
		hd_float e = in[(5*array+4)*size + j];
		return median5(a, b, c, d, e);
	}
};

struct median_scrunch5_beam_kernel
  : public thrust::unary_function<hd_float,hd_float> {
  const hd_float* in;
  const hd_size   in_size;
  const hd_size   out_size;
  median_scrunch5_beam_kernel(const hd_float* in_, hd_size in_size_, hd_size out_size_)
    : in(in_), in_size(in_size_), out_size(out_size_) {}
  inline __host__ __device__
  hd_float operator()(unsigned int o) const {
    // o is (output_sample * nbeam)
    hd_size beam = o / out_size; 
    // i is input sample
    hd_size i = (o % out_size) * 5;
    hd_size offset = (beam * in_size) + i;
    hd_float a = in[offset+0];
    hd_float b = in[offset+1];
    hd_float c = in[offset+2];
    hd_float d = in[offset+3];
    hd_float e = in[offset+4];
    return median5(a, b, c, d, e);
  }
};

struct median_beam_kernel
  : public thrust::unary_function<hd_float,hd_float> {
  const hd_float* in;
  const hd_size   size;
  median_beam_kernel(const hd_float* in_, hd_size size_)
    : in(in_), size(size_) {}
  inline __host__ __device__
  hd_float operator()(unsigned int o) const {
    hd_size offset = (o * size);
    if (size == 1)
      return in[offset+0];
    else if (size == 2)
      return 0.5f*(in[offset+0] + in[offset+1]);
    else if (size == 3)
      return median3 (in[offset+0], in[offset+1], in[offset+2]);
    else 
      return median4 (in[offset+0], in[offset+1], in[offset+2], in[offset+3]);
  }
};


hd_error median_filter3(const hd_float* d_in,
                        hd_size         count,
                        hd_float*       d_out)
{
	thrust::device_ptr<hd_float> d_out_begin(d_out);
	using thrust::make_counting_iterator;
	thrust::transform(make_counting_iterator<unsigned int>(0),
	                  make_counting_iterator<unsigned int>(count),
	                  d_out_begin,
	                  median_filter3_kernel(d_in, count));
	return HD_NO_ERROR;
}

hd_error median_filter5(const hd_float* d_in,
                        hd_size         count,
                        hd_float*       d_out)
{
	thrust::device_ptr<hd_float> d_out_begin(d_out);
	using thrust::make_counting_iterator;
	thrust::transform(make_counting_iterator<unsigned int>(0),
	                  make_counting_iterator<unsigned int>(count),
	                  d_out_begin,
	                  median_filter5_kernel(d_in, count));
	return HD_NO_ERROR;
}

hd_error median_scrunch3(const hd_float* d_in,
                         hd_size         count,
                         hd_float*       d_out)
{
	thrust::device_ptr<const hd_float> d_in_begin(d_in);
	thrust::device_ptr<hd_float>       d_out_begin(d_out);
	if( count == 1 ) {
		*d_out_begin = d_in_begin[0];
	}
	else if( count == 2 ) {
		*d_out_begin = 0.5f*(d_in_begin[0] + d_in_begin[1]);
	}
	else {
		// Note: Truncating here is necessary
		hd_size out_count = count / 3;
		using thrust::make_counting_iterator;
		thrust::transform(make_counting_iterator<unsigned int>(0),
		                  make_counting_iterator<unsigned int>(out_count),
		                  d_out_begin,
		                  median_scrunch3_kernel(d_in));
	}
	return HD_NO_ERROR;
}

hd_error median_scrunch5(const hd_float* d_in,
                         hd_size         count,
                         hd_float*       d_out)
{
	thrust::device_ptr<const hd_float> d_in_begin(d_in);
	thrust::device_ptr<hd_float>       d_out_begin(d_out);
	
	if( count == 1 ) {
		*d_out_begin = d_in_begin[0];
	}
	else if( count == 2 ) {
		*d_out_begin = 0.5f*(d_in_begin[0] + d_in_begin[1]);
	}
	else if( count == 3 ) {
		*d_out_begin = median3(d_in_begin[0],
		                       d_in_begin[1],
		                       d_in_begin[2]);
	}
	else if( count == 4 ) {
		*d_out_begin = median4(d_in_begin[0],
		                       d_in_begin[1],
		                       d_in_begin[2],
		                       d_in_begin[3]);
	}
	else {
		// Note: Truncating here is necessary
		hd_size out_count = count / 5;
		using thrust::make_counting_iterator;
		thrust::transform(make_counting_iterator<unsigned int>(0),
		                  make_counting_iterator<unsigned int>(out_count),
		                  d_out_begin,
		                  median_scrunch5_kernel(d_in));
	}
	return HD_NO_ERROR;
}

// Median-scrunches the corresponding elements from a collection of arrays
// Note: This cannot (currently) handle count not being a multiple of 3
hd_error median_scrunch3_array(const hd_float* d_in,
                               hd_size         array_size,
                               hd_size         count,
                               hd_float*       d_out)
{
	thrust::device_ptr<hd_float> d_out_begin(d_out);
	// Note: Truncating here is necessary
	hd_size out_count = count / 3;
	hd_size total     = array_size * out_count;
	using thrust::make_counting_iterator;
	thrust::transform(make_counting_iterator<unsigned int>(0),
	                  make_counting_iterator<unsigned int>(total),
	                  d_out_begin,
	                  median_scrunch3_array_kernel(d_in, array_size));
	return HD_NO_ERROR;
}

// Median-scrunches the corresponding elements from a collection of arrays
// Note: This cannot (currently) handle count not being a multiple of 5
hd_error median_scrunch5_array(const hd_float* d_in,
                               hd_size         array_size,
                               hd_size         count,
                               hd_float*       d_out)
{
	thrust::device_ptr<hd_float> d_out_begin(d_out);
	// Note: Truncating here is necessary
	hd_size out_count = count / 5;
	hd_size total     = array_size * out_count;
	using thrust::make_counting_iterator;
	//printf("%d %d\n",out_count,total);
	thrust::transform(make_counting_iterator<unsigned int>(0),
	                  make_counting_iterator<unsigned int>(total),
	                  d_out_begin,
	                  median_scrunch5_array_kernel(d_in, array_size));
	return HD_NO_ERROR;
}

// Median-scrunches the corresponding elements from a collection of arrays
// Note: This cannot (currently) handle count not being a multiple of 5
hd_error median_scrunch5_beam(const hd_float* d_in,
                              hd_size         beam_stride,
                              hd_size         nbeam,
                              hd_float*       d_out)
{
  thrust::device_ptr<hd_float> d_out_begin(d_out);
 
  if (beam_stride > 4)
  {
    // Note: Truncating here is necessary
    hd_size out_count = beam_stride / 5;
    hd_size total     = nbeam * out_count;
    //std::cout << "median_scrunch5_beam beam_stride=" << beam_stride << " out_count=" << out_count << " total=" << total << std::endl;
    using thrust::make_counting_iterator;
    thrust::transform(make_counting_iterator<unsigned int>(0),
                      make_counting_iterator<unsigned int>(total),
                      d_out_begin,
                      median_scrunch5_beam_kernel(d_in, beam_stride, out_count));
  }
  else
  {
    using thrust::make_counting_iterator;
    thrust::transform(make_counting_iterator<unsigned int>(0),
                      make_counting_iterator<unsigned int>(nbeam),
                      d_out_begin,
                      median_beam_kernel(d_in, beam_stride));

  }

  return HD_NO_ERROR;
}


template<typename T>
struct mean2_functor : public thrust::binary_function<T,T,T> {
	inline __host__ __device__
	T operator()(T a, T b) const { return (T)0.5 * (a+b); }
};

struct mean_scrunch2_array_kernel
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	const hd_size   size;
	mean_scrunch2_array_kernel(const hd_float* in_, hd_size size_)
		: in(in_), size(size_) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		hd_size array = i / size;
		hd_size j     = i % size;
		
		hd_float a = in[(2*array+0)*size + j];
		hd_float b = in[(2*array+1)*size + j];
		return (hd_float)0.5 * (a+b);
	}
};

// Note: This can operate 'in-place'
hd_error mean_filter2(const hd_float* d_in,
                      hd_size         count,
                      hd_float*       d_out)
{
	thrust::device_ptr<const hd_float> d_in_begin(d_in);
	thrust::device_ptr<hd_float>       d_out_begin(d_out);
	thrust::adjacent_difference(d_in_begin, d_in_begin+count,
	                            d_out_begin,
	                            mean2_functor<hd_float>());
	return HD_NO_ERROR;
}

hd_error mean_scrunch2_array(const hd_float* d_in,
                             hd_size         array_size,
                             hd_size         count,
                             hd_float*       d_out)
{
	thrust::device_ptr<hd_float> d_out_begin(d_out);
	// Note: Truncating here is necessary
	hd_size out_count = count / 2;
	hd_size total     = array_size * out_count;
	using thrust::make_counting_iterator;
	thrust::transform(make_counting_iterator<unsigned int>(0),
	                  make_counting_iterator<unsigned int>(total),
	                  d_out_begin,
	                  mean_scrunch2_array_kernel(d_in, array_size));
	return HD_NO_ERROR;
}

struct linear_stretch_functor
	: public thrust::unary_function<hd_float,hd_float> {
	const hd_float* in;
	hd_float        step;
	linear_stretch_functor(const hd_float* in_,
	                       hd_size in_count, hd_size out_count)
		: in(in_), step(hd_float(in_count-1)/(out_count-1)) {}
	inline __host__ __device__
	hd_float operator()(unsigned int i) const {
		hd_float     x = i * step;
		unsigned int j = x;
		return in[j] + ((x-j > 1e-5) ? (x-j)*(in[j+1]-in[j]) : 0.f);
	}
};

hd_error linear_stretch(const hd_float* d_in,
                        hd_size         in_count,
                        hd_float*       d_out,
                        hd_size         out_count)
{
	using thrust::make_counting_iterator;
	thrust::device_ptr<hd_float> d_out_begin(d_out);
	
	thrust::transform(make_counting_iterator<unsigned int>(0),
	                  make_counting_iterator<unsigned int>(out_count),
	                  d_out_begin,
	                  linear_stretch_functor(d_in, in_count, out_count));
	return HD_NO_ERROR;
}
