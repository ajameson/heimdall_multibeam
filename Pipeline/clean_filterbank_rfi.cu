#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "hd/clean_filterbank_rfi.h"
#include "hd/remove_baseline.h"
#include "hd/get_rms.h"
#include "hd/measure_bandpass.h"
#include "hd/matched_filter.h"
#include <stdio.h>

#include <vector>
#include <dedisp.h>
#ifdef HAVE_MPI
#include <mpi.h>
#endif
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>

// TESTING ONLY
#ifdef _DEBUG
#include "hd/write_time_series.h"
#endif

// A simple hashing function taken from Thrust's Monte Carlo example
inline __host__ __device__
unsigned int hash(unsigned int a) {
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

template<typename T>
struct abs_less_than : public thrust::unary_function<T,bool> {
  T thresh;
  abs_less_than(T thresh_) : thresh(thresh_) {}
  inline __host__ __device__
  bool operator()(T x) const {
    return fabs(x) < thresh;
  }
};

template<typename WordType>
struct zap_fb_rfi_functor : public thrust::unary_function<WordType,WordType> {
  // Note: Increasing this trades performance for accuracy
  enum { MAX_RESAMPLE_ATTEMPTS = 10 };
  const int*      mask;
  const WordType* in;
  unsigned int    stride;
  unsigned int    nbits;
  unsigned int    nsamps;
  unsigned int    max_resample_dist;
  WordType        bitmask;
  zap_fb_rfi_functor(const int* mask_, const WordType* in_,
                     unsigned int stride_, unsigned int nbits_,
                     unsigned int nsamps_, unsigned int max_resample_dist_)
    : mask(mask_), in(in_),
      stride(stride_), nbits(nbits_), bitmask((1<<nbits)-1),
      nsamps(nsamps_), max_resample_dist(max_resample_dist_) {}
  inline __host__ __device__
  WordType operator()(unsigned int i) const {
    // Lift the 1D index into 2D filterbank coords
    // Note: c is the word, not the channel
    unsigned int t = i / stride;
    unsigned int c = i % stride;
    WordType result;
    if( mask[t] ) {
      unsigned int seed = hash(i);
      // Create a random number engine for this thread
      // Note: This technique is succeptible to correlation between values
      //         A better, but slower, method is to use rng.discard( )
      // TODO: Consider passing a global seed (e.g., derived from the current
      //          time) in here to ensure good randomness.
      thrust::default_random_engine rng(seed);
      result = 0;
      // Iterate over channels in the word
      for( int k=0; k<sizeof(WordType)*8; k+=nbits ) {
        unsigned int min_t = t > max_resample_dist ?
          t - max_resample_dist : 0;
        unsigned int max_t = t < nsamps-1 - max_resample_dist ?
          t + max_resample_dist : nsamps-1;
        thrust::uniform_int_distribution<unsigned int> dist(min_t, max_t);
        unsigned int new_t = dist(rng);
        // Avoid replacing with another bad sample
        // Note: We must limit the number of attempts here for speed
        int attempts = 0;
        while( mask[new_t] && ++attempts < MAX_RESAMPLE_ATTEMPTS+1 ) {
          new_t = dist(rng);
        }
        
        WordType val = (in[new_t*stride + c] >> k) & bitmask;
        result |= val << k;
      }
    }
    else {
      // Return the input value unchanged
      result = in[i/*t*stride + c*/];
    }
    return result;
  }
};
template<typename WordType>
struct zap_narrow_rfi_functor : public thrust::unary_function<WordType,WordType> {
  // Note: Increasing this trades performance for accuracy
  enum { MAX_RESAMPLE_ATTEMPTS = 10 };
  WordType*       data;
  const float*    baseline;
  float           thresh;
  unsigned int    stride;
  unsigned int    nbits;
  unsigned int    nchans;
  unsigned int    max_resample_dist;
  WordType        bitmask;
  unsigned int    chans_per_word;
  zap_narrow_rfi_functor(WordType* data_, const float* baseline_,
                         float thresh_,
                         unsigned int stride_, unsigned int nbits_,
                         unsigned int nchans_, unsigned int max_resample_dist_)
    : data(data_), baseline(baseline_), thresh(thresh_),
      stride(stride_), nbits(nbits_), bitmask((1<<nbits)-1),
      nchans(nchans_), max_resample_dist(max_resample_dist_),
      chans_per_word(sizeof(WordType)*8/nbits) {}
  
  inline __host__ __device__
  WordType sample(unsigned int t, unsigned int c) const {
    unsigned int w = c / chans_per_word;
    unsigned int k = c % chans_per_word;
    return (data[t*stride + w] >> (k*nbits)) & bitmask;
  }
  
  inline __host__ __device__
  void operator()(unsigned int i) const {
    // Lift the 1D index into 2D filterbank coords
    unsigned int t = i / stride;
    unsigned int w = i % stride;
    WordType word = data[i];
    
    unsigned int seed = hash(i);
    // Create a random number engine for this thread
    // Note: This technique is succeptible to correlation between values
    //         A better, but slower, method is to use rng.discard( )
    // TODO: Consider passing a global seed (e.g., derived from the current
    //          time) in here to ensure good randomness.
    thrust::default_random_engine rng(seed);
    
    bool any_bad = false;
    // Iterate over channels in the word
    //for( int k=0; k<sizeof(WordType)*8; k+=nbits ) {
    for( int k=0; k<chans_per_word; ++k ) {
      unsigned int c = w + k;
      WordType val = (word >> (k*nbits)) & bitmask;
      if( fabs(val - baseline[c]) > thresh ) {
        any_bad = true;
        
        unsigned int min_c = c > max_resample_dist ?
          c - max_resample_dist : 0;
        unsigned int max_c = c < nchans-1 - max_resample_dist ?
          c + max_resample_dist : nchans-1;
        
        thrust::uniform_int_distribution<unsigned int> distn(min_c, max_c);
        unsigned int new_c = distn(rng);
        
        // Avoid replacing with another bad sample
        // Note: We must limit the number of attempts here for speed
        int attempts = 0;
        WordType new_val = sample(t, new_c);
        while( fabs(new_val - baseline[new_c]) > thresh &&
               ++attempts < MAX_RESAMPLE_ATTEMPTS+1 ) {
          new_c = distn(rng);
          new_val = sample(t, new_c);
        }
        // Replace the relevant bits
        word &= ~(bitmask << (k*nbits));
        word |= new_val << (k*nbits);
      }
    }
    if( any_bad ) {
      data[i] = word;
    }
  }
};

// Zaps the whole band for each masked time sample, replacing values with
//   others sampled randomly from nearby.
hd_error zap_filterbank_rfi(const int* h_mask, const hd_byte* h_in,
                            hd_size nsamps, hd_size nbits, hd_size nchans,
                            hd_size max_resample_dist,
                            hd_byte* h_out)
{
  unsigned int stride_bytes = nchans * nbits / 8;
  
  // Note: This type is used to optimise memory accesses
  //         It also sets the upper limit on nbits
  typedef unsigned int WordType;
  // TODO: Does this break things when nbits > 8 ?
  //typedef hd_byte WordType;
  // Note: This is the stride in words
  // TODO: This assumes the byte stride is a multiple of the word size,
  //         which may not be true.
  unsigned int stride = stride_bytes / sizeof(WordType);
  
  // TODO: Tidy this up. Could possibly pass device arrays rather than host.
  
  // Copy filterbank data to the device
  thrust::device_vector<WordType> d_in((WordType*)h_in,
                                       (WordType*)h_in + nsamps*stride);
  thrust::device_vector<WordType> d_out(nsamps*stride);
  thrust::device_vector<int>      d_mask(h_mask, h_mask+nsamps);
  WordType* d_in_ptr   = thrust::raw_pointer_cast(&d_in[0]);
  int*      d_mask_ptr = thrust::raw_pointer_cast(&d_mask[0]);
  thrust::transform(thrust::counting_iterator<unsigned int>(0),
                    thrust::counting_iterator<unsigned int>(nsamps*stride),
                    d_out.begin(),
                    zap_fb_rfi_functor<WordType>(d_mask_ptr, d_in_ptr,
                                                 stride, nbits,
                                                 nsamps, max_resample_dist));
  // Copy back to the host
  thrust::copy(d_out.begin(), d_out.end(),
               (WordType*)h_out);
  
  return HD_NO_ERROR;
}

template<typename T>
struct is_rfi : public thrust::unary_function<T, bool> {
  T thresh;
  is_rfi(T thresh_) : thresh(thresh_) {}
  inline __host__ __device__
  bool operator()(T x) const {
    return fabs(x) > thresh;
  }
};

template<typename T>
struct rfi_mask_functor : public thrust::binary_function<T,int,bool> {
  T thresh;
  rfi_mask_functor(T thresh_) : thresh(thresh_) {}
  inline __host__ __device__
  bool operator()(T x, int mask) const {
    return (fabs(x) > thresh) || mask;
  }
};

hd_error clean_filterbank_rfi(dedisp_plan    main_plan,
                              const hd_byte* h_in,
                              hd_size        nsamps,
                              hd_size        nbits,
                              hd_byte*       h_out,
                              int*           h_killmask,
                              hd_float       dm,
                              hd_float       dt,
                              hd_float       baseline_length,
                              hd_float       rfi_tol,
                              hd_size        rfi_min_beams,
                              hd_size        boxcar_max,
                              hd_size        nbeams)
{
  using thrust::counting_iterator;
  
  hd_error error;
  
  typedef hd_float out_type;
  std::vector<out_type>           h_raw_series;
  thrust::device_vector<hd_float> d_series;
  //thrust::host_vector<hd_float>   h_series;
  thrust::device_vector<hd_float> d_filtered;
  //thrust::host_vector<hd_float>   h_beams_series;
  //thrust::device_vector<hd_float> d_beams_series;
  thrust::device_vector<int>      d_filtered_rfi_mask;
  thrust::device_vector<int>      d_rfi_mask;
  thrust::host_vector<int>        h_rfi_mask;
  
  hd_size nchans = dedisp_get_channel_count(main_plan);
  
  // TODO: Any way to avoid having to use this?
  thrust::host_vector<hd_byte> h_in_copy;
  
  typedef unsigned int WordType;
  hd_size stride = nchans * nbits/8 / sizeof(WordType);
  
  // TODO: Any way to avoid having to use this?
  thrust::device_vector<WordType> d_in((WordType*)h_in,(WordType*)h_in + nsamps*stride);
  
  WordType* d_in_ptr = thrust::raw_pointer_cast(&d_in[0]);
  
  thrust::device_vector<hd_float> d_bandpass(nchans);
  hd_float* d_bandpass_ptr = thrust::raw_pointer_cast(&d_bandpass[0]);
  
  // Narrow-band RFI is not an issue when nbits is small
  // Note: Small nbits can actually cause this excision code to fail
  if( nbits > 4 && false) {
    // Narrow-band RFI excision
    // ------------------------
    // TODO: Any motivation for this?
    //       Make it a parameter?
    hd_size max_chan_resample_dist = nchans / 60;
    
    // We loop over gulps of nsamps_smooth samples so that each one
    //   gets its own bandpass measurement.
    // TODO: Should this be halved? (Note: adds 25% to total cleaning time)
    hd_size nsamps_smooth = hd_size(baseline_length / (1 * dt));
    for( hd_size g=0; g<nsamps; g+=nsamps_smooth ) {
      hd_size nsamps_gulp = std::min(nsamps_smooth, nsamps-g);
      std::cerr << "g=" << g << " meansure_bandpass" << std::endl;
 
      // Measure the bandpass
      hd_float rms = 0;
      measure_bandpass((hd_byte*)(d_in_ptr + g*stride),
                       nsamps_gulp, nchans, nbits,
                       d_bandpass_ptr, &rms);
      
      /* try {}
      catch(thrust::system_error &e) 
		{
			printf("Failed4\n");
			exit(-1);
		}	
*/
      	    
      std::cerr << "g=" << g << " zap_narrow_rfi_functor" << std::endl;

      zap_narrow_rfi_functor<WordType> zapit(d_in_ptr,
                                             d_bandpass_ptr,
                                             rfi_tol*rms,
                                             stride, nbits, nchans,
                                             max_chan_resample_dist);
      
      // Zap narrow-band RFI
      counting_iterator<unsigned int> begin(g*stride);
      counting_iterator<unsigned int> end((g+nsamps_gulp)*stride);
      thrust::for_each(begin, end,
                       zapit);
    }
    
    h_in_copy.resize(nsamps*stride*sizeof(WordType));
    thrust::copy(d_in.begin(), d_in.end(),
                 (WordType*)&h_in_copy[0]);
  }
  else {
    h_in_copy.assign(h_in, h_in+nsamps*nchans*nbits/8);
  }
  // ------------------------
  
  // Broad-band RFI excision
  // First, dedisperse at the given DM
  // ---------------------------------
  dedisp_error derror;
  // Create a new plan for the zero-DM dedispersion
  dedisp_float f0 = dedisp_get_f0(main_plan);
  dedisp_float df = dedisp_get_df(main_plan);
  dedisp_plan plan;
  derror = dedisp_create_plan(&plan, nchans, dt, f0, df);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  
  derror = dedisp_disable_adaptive_dt(plan);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  derror = dedisp_set_dm_list(plan, &dm, 1);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  hd_size max_delay       = dedisp_get_max_delay(plan);

  hd_size beam_stride     = nsamps / nbeams;
  hd_size nsamps_computed = nsamps - max_delay;
  hd_size beam_nsamps     = beam_stride - max_delay;
  
  h_raw_series.resize(nsamps_computed);
  
  unsigned flags = DEDISP_USE_DEFAULT;
  const dedisp_byte* in        = (const dedisp_byte*)&h_in_copy[0];
  dedisp_byte*       out       = (dedisp_byte*)&h_raw_series[0];
  hd_size            out_nbits = sizeof(out_type)*8;
  derror = dedisp_execute(plan, nsamps,
                          in, nbits,// in_stride,
                          out, out_nbits,// out_stride,
                          //gulp_dm, dm_gulp_size,
                          flags);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  dedisp_destroy_plan(plan);
  // ---------------------------------
  
  // Then baseline and normalise the time series
  // -------------------------------------------
  // Copy to the device and convert to floats
  d_series = h_raw_series;
  // Remove the baseline
  hd_size nsamps_smooth = hd_size(baseline_length / (2 * dt));
  hd_float* d_series_ptr = thrust::raw_pointer_cast(&d_series[0]);
  
#ifdef _DEBUG
  write_device_time_series(d_series_ptr, nsamps_computed,
                           dt, "dm0_dedispersed.tim");
#endif
  
  RemoveBaselinePlan baseline_remover;
  error = baseline_remover.exec(d_series_ptr, beam_stride, beam_nsamps, nsamps_smooth, nbeams);
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }
  
#ifdef _DEBUG
  write_device_time_series(d_series_ptr, nsamps_computed,
                           dt, "dm0_baselined.tim");
#endif
  
  thrust::device_vector<hd_float> d_beam_rms;
  d_beam_rms.resize(nbeams);
  hd_float* beam_rms = thrust::raw_pointer_cast(&d_beam_rms[0]);

  // multibeam normalisation, first get the RMS of each beam
  GetRMSPlanMB rms_getter;
  error = rms_getter.exec_multibeam(d_series_ptr, beam_rms, beam_stride, beam_nsamps, nbeams);
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }

  // normalise each beam with each RMS
  error = normalise_multibeam (d_series_ptr, beam_rms, beam_stride, nsamps_computed, nbeams);
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }

#ifdef _DEBUG
  write_device_time_series(d_series_ptr, nsamps_computed,
                           dt, "dm0_normalised.tim");
#endif
  // -------------------------------------------
  
  // Do a simple sigma cut to identify RFI
  // -------------------------------------
  d_rfi_mask.resize(nsamps_computed, 0);
  
  d_filtered_rfi_mask.resize(nsamps_computed, 0);
  int* d_filtered_rfi_mask_ptr =
    thrust::raw_pointer_cast(&d_filtered_rfi_mask[0]);
  
  // Create an RFI mask for this filter
  thrust::transform(d_series.begin(), d_series.end(),
                    d_rfi_mask.begin(),
                    is_rfi<hd_float>(rfi_tol));
  
  // Note: The filtered output is shorter by boxcar_max samps
  //         and offset by boxcar_max/2 samps.
  d_filtered.resize(nsamps_computed + 1 - boxcar_max);
  hd_float* d_filtered_ptr = thrust::raw_pointer_cast(&d_filtered[0]);
  MatchedFilterPlan<hd_float> filter_plan;
  filter_plan.prep(d_series_ptr, nsamps_computed, boxcar_max);
  
  for( hd_size filter_width=1; filter_width<=boxcar_max;
       filter_width*=2 ) {
    
    // Apply the matched filter
    // Note: The filtered output is shorter by boxcar_max samps
    //         and offset by (boxcar_max-1)/2+1 samps.
    filter_plan.exec(d_filtered_ptr, filter_width);
    
    // Normalise the filtered time series (RMS ~ sqrt(time))
    thrust::constant_iterator<hd_float> 
      norm_val_iter(1.0 / sqrt((hd_float)filter_width));
    thrust::transform(d_filtered.begin(),
                      d_filtered.end(),
                      norm_val_iter,
                      d_filtered.begin(),
                      thrust::multiplies<hd_float>());
    
    //hd_size filter_offset = (boxcar_max-1)/2+1;
    hd_size filter_offset = boxcar_max / 2;
    
    // Create an RFI mask for this filter
    thrust::transform(d_filtered.begin(), d_filtered.end(),
                      d_filtered_rfi_mask.begin() + filter_offset,
                      is_rfi<hd_float>(rfi_tol));
    
    // Filter the RFI mask
    // Note: This ensures we zap all samples contributing to the peak
    MatchedFilterPlan<int> mask_filter_plan;
    mask_filter_plan.prep(d_filtered_rfi_mask_ptr, nsamps_computed,
                          boxcar_max);
    mask_filter_plan.exec(d_filtered_rfi_mask_ptr + filter_offset,
                          filter_width);
    
    // Merge the filtered mask with the global mask
    thrust::transform(d_rfi_mask.begin(), d_rfi_mask.end(),
                      d_filtered_rfi_mask.begin(),
                      d_rfi_mask.begin(),
                      thrust::logical_or<int>());
  }
  h_rfi_mask = d_rfi_mask;
  // -------------------------------------
  
  // Finally, apply the mask to zap RFI in the filterbank
  error = zap_filterbank_rfi(&h_rfi_mask[0],
                             &h_in_copy[0],
                             nsamps_computed,
                             nbits,
                             nchans,
                             // TODO: This is somewhat arbitrary
                             nsamps_smooth/4,
                             &h_out[0]);
  if( error != HD_NO_ERROR ) {
    return error;
  }
  
#ifdef _DEBUG
  write_host_time_series((unsigned int*) &h_in_copy[0], nsamps_computed, nbits, dt, "dm0_dirty.tim");
  write_host_time_series((unsigned int*) &h_out[0], nsamps_computed, nbits, dt, "dm0_cleaned.tim");
#endif

  return HD_NO_ERROR;
}

hd_error apply_manual_killmasks (dedisp_plan    main_plan,
                                 int*           h_killmask,
                                 unsigned int num_channel_zaps,
                                 hd_range_t * channel_zaps)
{
  hd_size nchans = dedisp_get_channel_count(main_plan);
  for (unsigned i=0; i<num_channel_zaps; i++)
  {
    for (unsigned j=channel_zaps[i].start; j<channel_zaps[i].end; j++)
    {
      if (j < nchans)
        h_killmask[j] = 0;
    }
  }
  return HD_NO_ERROR;
}
