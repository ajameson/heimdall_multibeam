#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "hd/label_candidate_clusters.h"
#include "hd/are_coincident.cuh"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/binary_search.h>
#include <thrust/count.h>

/*
// Lexicographically projects 3D integer coordinates onto a 1D coordinate
// Also applies an offset and performs boundary clamping
template<typename T>
struct project_coords_functor : public thrust::unary_function<void,T> {
	int width;
	int height;
	int depth;
	int stride_x;
	int stride_y;
	int stride_z;
	int dx;
	int dy;
	int dz;
	project_coords_functor() {}
	project_coords_functor(int w, int h, int d, int rad,
	                       int dx_=0, int dy_=0, int dz_=0)
		: width(w), height(h), depth(d),
		  stride_x(1),
		  stride_y(w),//+2*rad),
		  stride_z(h*(w)),//+2*rad)),
		  dx(dx_), dy(dy_), dz(dz_) {}
	
	template<typename CoordTuple>
	inline __host__ __device__
	T operator()(CoordTuple xyz) const {
		//int x = thrust::get<0>(xyz) + dx;
		// HACK TESTING
		int filter_width = 1 << thrust::get<1>(xyz);
		int x = thrust::get<0>(xyz) + dx * filter_width;
		
		x = min(max(x,0),width-1);
		int y = (int)thrust::get<1>(xyz) + dy;
		int z = (int)thrust::get<2>(xyz) + dz;
		// We must be careful around the borders
		if( z < 0 ) {
			x = 0;
		}
		else if( z > depth-1 ) {
			x = width-1;
		}
		else if( y < 0 ) {
			x = 0;
		}
		else if( y > height-1 ) {
			x = width-1;
		}
		y = min(max(y,0),height-1);
		z = min(max(z,0),depth-1);
		return x * stride_x + y * stride_y + z * stride_z;
	}
};

// Returns the minimum over a specified range of elements
template<typename ValueType, typename SizeType>
struct range_min_functor : public thrust::binary_function<void,void,ValueType> {
	const ValueType* data;
	range_min_functor(const ValueType* data_) : data(data_) {}
	
	template<typename BeginEndTuple>
	inline __host__ __device__
	ValueType operator()(ValueType init, BeginEndTuple begin_end) const {
		SizeType begin = thrust::get<0>(begin_end);
		SizeType end   = thrust::get<1>(begin_end);
		ValueType result = init;
		for( SizeType i=begin; i<end; ++i ) {
			// TODO: What's with the stupid min( ) overloads?
			//result = min((long long)result, (long long)data[i]);
			result = (result <= data[i]) ? result : data[i];
		}
		return result;
	}
};
*/
__device__ unsigned int d_counter;

// Finds the root of a chain of equivalent labels
//   E.g., 3->1, 4->3, 8->4, 5->8 => [1,3,4,5,8]->1
// TODO: It would be quite interesting to study the behaviour of this
//         algorithm/implementation in more detail.
template<typename T>
struct trace_equivalency_chain {
	T* new_labels;
	trace_equivalency_chain(T* new_labels_) : new_labels(new_labels_) {}
	inline /*__host__*/ __device__
	void operator()(unsigned int old_label) const {
		T cur_label = old_label;
		while( new_labels[cur_label] != cur_label ) {
			cur_label = new_labels[cur_label];
			//new_labels[old_label] = cur_label;
			// TESTING TODO: See if/how this varies if we write
			//                 new_labels[old_label] each iteration vs.
			//                 only at the end (see commented line below).
			//               It appears to make only 10-20% difference
			atomicAdd(&d_counter, 1);
		}
		new_labels[old_label] = cur_label;
		
		/*
		T j = i;
		while( new_labels[i] != j ) {
			new_labels[i] = new_labels[j];
		}
		
		
		T j = i;
		T new_label = new_labels[i] = new_labels[j];
		while( new_label != j ) {
			j = new_label;
			//new_label = new_labels[j];
			new_label = new_labels[i] = new_labels[j];
		}
		// Note: This written value may subsequently be read by another thread,
		//         which should improve the speed of the algorithm by exploiting
		//         already-computed results.
		new_labels[i] = new_label;
		*/
	}
};

struct cluster_functor {
	hd_size  count;
	const hd_size* d_samp_inds;
	const hd_size* d_begins;
	const hd_size* d_ends;
	const hd_size* d_filters;
	const hd_size* d_dms;
	hd_size* d_labels;
	hd_size  time_tol;
	hd_size  filter_tol;
	hd_size  dm_tol;
	hd_size  nsamps_beam;
	
	cluster_functor(hd_size count_,
	                const hd_size* d_samp_inds_,
	                const hd_size* d_begins_, const hd_size* d_ends_,
	                const hd_size* d_filters_, const hd_size* d_dms_,
	                hd_size* d_labels_,
	                hd_size time_tol_, hd_size filter_tol_, hd_size dm_tol_,
                  hd_size nsamps_beam_)
		: count(count_),
		  d_samp_inds(d_samp_inds_),
		  d_begins(d_begins_), d_ends(d_ends_),
		  d_filters(d_filters_), d_dms(d_dms_),
		  d_labels(d_labels_),
		  time_tol(time_tol_),
      filter_tol(filter_tol_),
      dm_tol(dm_tol_),
      nsamps_beam(nsamps_beam_) {}
	
	inline __host__ __device__
	void operator()(unsigned int i) {
		hd_size samp_i   = d_samp_inds[i];
		hd_size beam_i   = d_samp_inds[i] / nsamps_beam;
		hd_size filter_i = d_filters[i];
		hd_size dm_i     = d_dms[i];

		// TODO: This would be much faster using shared mem like in nbody
		for( unsigned int j=0; j<count; ++j ) {
			if( j == i ) {
				continue;
			}
			hd_size samp_j   = d_samp_inds[j];
			hd_size beam_j   = d_samp_inds[j] / nsamps_beam;
			hd_size filter_j = d_filters[j];
			hd_size dm_j     = d_dms[j];
			if( are_coincident_beam(samp_i, samp_j,
                              beam_i, beam_j,
                              filter_i, filter_j,
                              dm_i, dm_j,
                              time_tol, filter_tol, dm_tol) ) {
				// Re-label as the minimum of the two
				d_labels[i] = min((int)d_labels[i], (int)d_labels[j]);
			}
		}
	}
};

// Finds components of the given list that are connected in time, filter and DM
// Note: merge_dist is the distance in time up to which components are connected
// Note: Merge distances in filter and DM space are currently fixed at 1
// TODO: Consider re-naming the *_count args to *_max
hd_error label_candidate_clusters(hd_size            count,
                                  ConstRawCandidates d_cands,
                                  hd_size            time_tol,
                                  hd_size            filter_tol,
                                  hd_size            dm_tol,
                                  hd_size            nsamps_beam,
                                  hd_size*           d_labels,
                                  hd_size*           label_count)
{
	/*
	  def within_range(bi, ei, bj, ej, tol):
	      return bi <= ej+tol and bj <= ei+tol;
	      //return ej - bi >= 0 and ei - bj >= 0;
	      //return ej - bi >= -tol and ei - bj >= -tol;
	  
	  for ci in candidates:
	      for cj in candidates:
	          if ci == cj:
	              continue
	          if( abs(ci.dm_ind - cj.dm_ind) <= dm_ind_tol &&
	              abs(ci.filter_ind - cj.filter_ind) <= filter_ind_tol &&
	              within_range(ci.begin,ci.end,cj.begin,cj.end,time_tol) ):
	              ci.new_label = min(ci.new_label, cj.new_label);
	 */

	using thrust::make_counting_iterator;
	
	thrust::device_ptr<hd_size> d_labels_begin(d_labels);
	thrust::sequence(d_labels_begin, d_labels_begin+count);
	
	// This just does a brute-force O(N^2) search for neighbours and
	//   re-labels as the minimum label over neighbours.
	thrust::for_each(make_counting_iterator<unsigned int>(0),
	                 make_counting_iterator<unsigned int>(count),
	                 cluster_functor(count,
	                                 d_cands.inds,
	                                 d_cands.begins,
	                                 d_cands.ends,
	                                 d_cands.filter_inds,
	                                 d_cands.dm_inds,
	                                 d_labels,
	                                 time_tol,
	                                 filter_tol,
	                                 dm_tol,
                                   nsamps_beam));

	/*
	using thrust::make_transform_iterator;
	using thrust::make_zip_iterator;
	using thrust::make_counting_iterator;
	
	typedef thrust::device_ptr<const hd_size> const_coord_iterator;
	typedef thrust::device_ptr<hd_size>             coord_iterator;
	
	const_coord_iterator d_begins_begin(d_begins);
	const_coord_iterator d_ends_begin(d_ends);
	//const_coord_iterator d_beams_begin(d_beams);
	const_coord_iterator d_filters_begin(d_filter_inds);
	const_coord_iterator d_dms_begin(d_dm_inds);
	coord_iterator       d_labels_begin(d_labels);
	
	typedef thrust::device_vector<hd_size> coord_vector;
	coord_vector d_new_labels(count);
	coord_vector d_neib_begins(count);
	coord_vector d_neib_ends(count);
	
	thrust::sequence(d_labels_begin, d_labels_begin+count);
	
	typedef thrust::zip_iterator<thrust::tuple<
	                             const_coord_iterator,
	                             const_coord_iterator,
	                             const_coord_iterator> > coords_iterator;
	
	coords_iterator begin_coords(thrust::make_tuple(d_begins_begin,
	                                                d_filters_begin,
	                                                d_dms_begin));
	coords_iterator end_coords(thrust::make_tuple(d_ends_begin,
	                                              d_filters_begin,
	                                              d_dms_begin));
	
	project_coords_functor<hd_size> project_coords(time_count,
	                                               filter_count,
	                                               dm_count,
	                                               merge_dist);
	project_coords_functor<hd_size> project_offset_coords;
	
	hd_size search_count = 14;
	// Note: This list could be expanded to connect components over greater
	//         dists in filter and DM space (currently 1).
	//int     search_offsets[][2] = {{-1,0}, {0,-1}, {-1,-1}};
	int     search_offsets[][2] = {{-1,0}, {1,0},
	                               {-1,-1}, {0,-1}, {1,-1},
	                               {-1,-2}, {0,-2}, {1,-2},
	                               {-1,-3}, {0,-3}, {1,-3},
	                               {-1,-4}, {0,-4}, {1,-4}};
	for( hd_size i=0; i<search_count; ++i ) {
		//std::cout << "Searching around offset "
		//          << search_offsets[i][0] << ", "
		//          << search_offsets[i][1] << std::endl;
		// Find the beginning of each element's neighbours
		project_offset_coords =
			project_coords_functor<hd_size>(time_count,
			                                filter_count,
			                                dm_count,
			                                merge_dist,
			                                -(int)merge_dist+1,
			                                search_offsets[i][0],
			                                search_offsets[i][1]);
		thrust::lower_bound(make_transform_iterator(end_coords,
		                                            project_coords),
		                    make_transform_iterator(end_coords,
		                                            project_coords)+count,
		                    make_transform_iterator(begin_coords,
		                                            project_offset_coords),
		                    make_transform_iterator(begin_coords,
		                                            project_offset_coords)+count,
		                    d_neib_begins.begin());
		//std::cout << "neib_begins:" << std::endl;
		//thrust::copy(d_neib_begins.begin(), d_neib_begins.end(),
		//             std::ostream_iterator<hd_size>(std::cout, "\t"));
		//std::cout << std::endl;
		
		// Find the end of each element's neighbours
		project_offset_coords =
			project_coords_functor<hd_size>(time_count+merge_dist*2,
			                                filter_count,
			                                dm_count,
			                                merge_dist,
			                                +merge_dist,
			                                search_offsets[i][0],
			                                search_offsets[i][1]);
		thrust::upper_bound(make_transform_iterator(begin_coords,
		                                            project_coords),
		                    make_transform_iterator(begin_coords,
		                                            project_coords)+count,
		                    make_transform_iterator(end_coords,
		                                            project_offset_coords),
		                    make_transform_iterator(end_coords,
		                                            project_offset_coords)+count,
		                    d_neib_ends.begin());
		//std::cout << "neib_ends:" << std::endl;
		//thrust::copy(d_neib_ends.begin(), d_neib_ends.end(),
		//             std::ostream_iterator<hd_size>(std::cout, "\t"));
		//std::cout << std::endl;
		
		// Now find the minimum label over each element's neighbours
		thrust::transform(d_labels_begin, d_labels_begin+count,
		                  make_zip_iterator(thrust::make_tuple(d_neib_begins.begin(),
		                                                       d_neib_ends.begin())),
		                  d_new_labels.begin(),
		                  range_min_functor<hd_size,hd_size>(d_labels));
		//std::cout << "new_labels:" << std::endl;
		//thrust::copy(d_new_labels.begin(), d_new_labels.end(),
		//             std::ostream_iterator<hd_size>(std::cout, "\t"));
		//std::cout << std::endl;
		
		// Copy the latest labels to the 
		thrust::copy(d_new_labels.begin(), d_new_labels.end(),
		             d_labels_begin);
	}
	*/
	// Finally, trace equivalency chains to find the final labels
	// Note: This is a parallel version of this algorithm that may not be
	//         as efficient as the sequential version but should win out
	//         in overall speed.


	unsigned int* d_counter_address;
	hipGetSymbolAddress((void**)&d_counter_address, d_counter);
	thrust::device_ptr<unsigned int> d_counter_ptr(d_counter_address);
	*d_counter_ptr = 0;

	thrust::for_each(make_counting_iterator<unsigned int>(0),
	                 make_counting_iterator<unsigned int>(count),
	                 trace_equivalency_chain<hd_size>(d_labels));

	//std::cout << "Total chain iterations: " << *d_counter_ptr << std::endl;
	
	// Finally we do a quick count of the number of unique labels
	//   This is efficiently achieved by checking where new labels are
	//     unchanged from their original values (i.e., where d_labels[i] == i)
	thrust::device_vector<int> d_label_roots(count);
	thrust::transform(d_labels_begin, d_labels_begin+count,
	                  make_counting_iterator<hd_size>(0),
	                  d_label_roots.begin(),
	                  thrust::equal_to<hd_size>());
	*label_count = thrust::count_if(d_label_roots.begin(),
	                                d_label_roots.end(),
	                                thrust::identity<hd_size>());
	
	return HD_NO_ERROR;
}
