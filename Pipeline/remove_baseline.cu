#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "hd/remove_baseline.h"
#include "hd/median_filter.h"
//#include "hd/write_time_series.h"

#include <thrust/device_vector.h>


class RemoveBaselinePlan_impl {
	thrust::device_vector<hd_float> buf1;
	thrust::device_vector<hd_float> buf2;
	thrust::device_vector<hd_float> baseline;
public:
	hd_error exec(hd_float* d_data, hd_size beam_stride, hd_size beam_count,
	              hd_size smooth_radius, hd_size nbeams) {
		
		thrust::device_ptr<hd_float> d_data_begin(d_data);
	
		// This algorithm works by scrunching the data down to a time resolution
		//   representative of the desired smoothing length and then stretching
		//   it back out again. The scrunching is done using the median-of-5
		//   to ensure robustness against outliers (e.g., strong RFI spikes).
    hd_size dm_delay    = beam_stride - beam_count;

    // Note: This parameter allows tuning to match the smoothing length
    //         of the original iterative-clipping algorithm.
    hd_float oversample = 2;
    // Find the desired time resolution
    hd_size  sample_count = (hd_size)(oversample * hd_float(beam_count)/(2*smooth_radius) + 0.5);
    if( sample_count == 0 ) {
      // Too few samples, no need to baseline
      return HD_NO_ERROR;
    }

    // As we will use median-of-5, round to sample_count times a power of five
    hd_size nscrunches  = (hd_size)(log(beam_count/sample_count)/log(5.));
    hd_size count_round = pow(5.,nscrunches)*sample_count;

    //std::cerr << "beam_stride=" << beam_stride << " count=" << count << " nbeams=" << nbeams
    //          << " sample_count=" << sample_count << " nscrunches=" << nscrunches 
    //          << " count_round=" << count_round << std::endl;

    buf1.resize(count_round);
    buf2.resize(count_round/5);
    hd_float* buf1_ptr = thrust::raw_pointer_cast(&buf1[0]);
    hd_float* buf2_ptr = thrust::raw_pointer_cast(&buf2[0]);

    baseline.resize(beam_stride);
    hd_float* baseline_ptr = thrust::raw_pointer_cast(&baseline[0]);
    
    for (hd_size ibeam=0; ibeam<nbeams; ibeam++)
    {
      hd_size beam_offset = ibeam * beam_stride;
      hd_size count = beam_stride;
      if (ibeam == (nbeams - 1))
      {
        count = beam_count;
      }

      // First we re-sample to the rounded size
      linear_stretch(d_data + beam_offset, beam_count, buf1_ptr, count_round);
    
      // Then we median scrunch until we reach the sample size
      for( hd_size size=count_round; size>sample_count; size/=5 ) {
        median_scrunch5(buf1_ptr, size, buf2_ptr);
        std::swap(buf1_ptr, buf2_ptr);
      }
      // Note: Output is now at buf1_ptr
      thrust::device_ptr<hd_float> buf1_begin(buf1_ptr);
      thrust::device_ptr<hd_float> buf2_begin(buf2_ptr);
    
      // Then we need to extrapolate the ends
      linear_stretch(buf1_ptr, sample_count, buf2_ptr+1, sample_count*2);
      buf2_begin[0]                = 2*buf2_begin[1] - buf2_begin[2];
      buf2_begin[sample_count*2+1] = (2*buf2_begin[sample_count*2] -
                                      buf2_begin[sample_count*2-1]);
    
      // And finally we stretch back to the original length
      linear_stretch(buf2_ptr, sample_count*2+2, baseline_ptr, count);
    
      // TESTING
      // write_device_time_series(d_data, count, 1.f, "pre_baseline.tim");
      // write_device_time_series(baseline_ptr, count, 1.f, "thebaseline.tim");
    
      // Now we just subtract it off
      thrust::transform(d_data_begin + beam_offset, d_data_begin+beam_offset+count,
                        baseline.begin(),
                        d_data_begin + beam_offset,
                        thrust::minus<hd_float>());
    
      // TESTING
      // write_device_time_series(d_data, count, 1.f, "post_baseline.tim");
    }

    // write_device_time_series(d_data, (beam_stride*nbeams)-dm_delay, 1.f, "post_baselines.tim");

    return HD_NO_ERROR;
	}
};

// Public interface (wrapper for implementation)
RemoveBaselinePlan::RemoveBaselinePlan()
	: m_impl(new RemoveBaselinePlan_impl) {}
hd_error RemoveBaselinePlan::exec(hd_float* d_data, hd_size beam_stride, hd_size count,
                                  hd_size smooth_radius,
                                  hd_size nbeams) {
	return m_impl->exec(d_data, beam_stride, count, smooth_radius, nbeams);
}
