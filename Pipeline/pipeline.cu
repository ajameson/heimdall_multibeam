#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include <vector>
#include <memory>
#include <iostream>
using std::cout;
using std::cerr;
using std::endl;
#include <sstream>
#include <iomanip>
#include <string>
#include <fstream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using thrust::host_vector;
using thrust::device_vector;
#include <thrust/version.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/gather.h>

#include "hd/pipeline.h"
#include "hd/maths.h"
#include "hd/clean_filterbank_rfi.h"

#include "hd/remove_baseline.h"
#include "hd/matched_filter.h"
#include "hd/get_rms.h"
#include "hd/find_giants.h"
#include "hd/label_candidate_clusters.h"
#include "hd/merge_candidates.h"

#include "hd/DataSource.h"
#include "hd/ClientSocket.h"
#include "hd/SocketException.h"
#include "hd/stopwatch.h"         // For benchmarking
//#include "write_time_series.h" // For debugging

#include <dedisp.h>

#define HD_BENCHMARK

#ifdef HD_BENCHMARK
  void start_timer(Stopwatch& timer) { timer.start(); }
  void stop_timer(Stopwatch& timer) { hipDeviceSynchronize(); timer.stop(); }
#else
  void start_timer(Stopwatch& timer) { }
  void stop_timer(Stopwatch& timer) { }
#endif // HD_BENCHMARK

#include <utility> // For std::pair
template<typename T, typename U>
std::pair<T&,U&> tie(T& a, U& b) { return std::pair<T&,U&>(a,b); }

struct hd_pipeline_t {
  hd_params   params;
  dedisp_plan dedispersion_plan;
  //MPI_Comm    communicator;

  // Memory buffers used during pipeline execution
  std::vector<hd_byte>    h_clean_filterbank;
  host_vector<hd_byte>    h_dm_series;
  device_vector<hd_float> d_time_series;
  device_vector<hd_float> d_filtered_series;
};

hd_error allocate_gpu(const hd_pipeline pl) {
  // TODO: This is just a simple proc-->GPU heuristic to get us started
  int gpu_count;
  hipGetDeviceCount(&gpu_count);
  //int proc_idx;
  //MPI_Comm comm = pl->communicator;
  //MPI_Comm_rank(comm, &proc_idx);
  int proc_idx = pl->params.beam;
  int gpu_idx = pl->params.gpu_id;
  
  hipError_t cerror = hipSetDevice(gpu_idx);
  if( cerror != hipSuccess ) {
    cerr << "Could not setCudaDevice to " << gpu_idx << ": " << hipGetErrorString(cerror) <<  endl;
    return throw_cuda_error(cerror);
  }
  
  if( pl->params.verbosity >= 1 ) {
    cout << "Process " << proc_idx << " using GPU " << gpu_idx << endl;
  }
  
  if( !pl->params.yield_cpu ) {
    if( pl->params.verbosity >= 2 ) {
      cout << "\tProcess " << proc_idx << " setting CPU to spin" << endl;
    }
    cerror = hipSetDeviceFlags(hipDeviceScheduleSpin);
    if( cerror != hipSuccess ) {
      return throw_cuda_error(cerror);
    }
  }
  else {
    if( pl->params.verbosity >= 2 ) {
      cout << "\tProcess " << proc_idx << " setting CPU to yield" << endl;
    }
    // Note: This Yield flag doesn't seem to work properly.
    //   The BlockingSync flag does the job, although it may interfere
    //     with GPU/CPU overlapping (not currently used).
    //cerror = hipSetDeviceFlags(hipDeviceScheduleYield);
    cerror = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if( cerror != hipSuccess ) {
      return throw_cuda_error(cerror);
    }
  }
  
  return HD_NO_ERROR;
}

unsigned int get_filter_index(unsigned int filter_width) {
  // This function finds log2 of the 32-bit power-of-two number v
  unsigned int v = filter_width;
  static const unsigned int b[] = {0xAAAAAAAA, 0xCCCCCCCC, 0xF0F0F0F0, 
                                   0xFF00FF00, 0xFFFF0000};
  register unsigned int r = (v & b[0]) != 0;
  for( int i=4; i>0; --i) {
    r |= ((v & b[i]) != 0) << i;
  }
  return r;
}

hd_error hd_create_pipeline(hd_pipeline* pipeline_, hd_params params) {
  *pipeline_ = 0;
  
  // Note: We use a smart pointer here to automatically clean up after errors
  typedef std::auto_ptr<hd_pipeline_t> smart_pipeline_ptr;
  smart_pipeline_ptr pipeline = smart_pipeline_ptr(new hd_pipeline_t());
  if( !pipeline.get() ) {
    return throw_error(HD_MEM_ALLOC_FAILED);
  }
  
  pipeline->params = params;
  
  if( params.verbosity >= 2 ) {
    cout << "\tAllocating GPU..." << endl;
  }
  
  hd_error error = allocate_gpu(pipeline.get());
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }
  
  if( params.verbosity >= 3 ) {
    cout << "nchans = " << params.nchans << endl;
    cout << "dt     = " << params.dt << endl;
    cout << "f0     = " << params.f0 << endl;
    cout << "df     = " << params.df << endl;
  }
  
  if( params.verbosity >= 2 ) {
    cout << "\tCreating dedispersion plan..." << endl;
  }
  
  dedisp_error derror;
  derror = dedisp_create_plan(&pipeline->dedispersion_plan,
                              params.nchans, params.dt,
                              params.f0, params.df);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  // TODO: Consider loading a pre-generated DM list instead for flexibility
  derror = dedisp_generate_dm_list(pipeline->dedispersion_plan,
                                   pipeline->params.dm_min,
                                   pipeline->params.dm_max,
                                   pipeline->params.dm_pulse_width,
                                   pipeline->params.dm_tol);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  
  if( pipeline->params.use_scrunching ) {
    derror = dedisp_enable_adaptive_dt(pipeline->dedispersion_plan,
                                       pipeline->params.dm_pulse_width,
                                       pipeline->params.scrunch_tol);
    if( derror != DEDISP_NO_ERROR ) {
      return throw_dedisp_error(derror);
    }
  }
  
  *pipeline_ = pipeline.release();
  
  if( params.verbosity >= 2 ) {
    cout << "\tInitialisation complete." << endl;
  }
  
  if( params.verbosity >= 1 ) {
    cout << "Using Thrust v"
         << THRUST_MAJOR_VERSION << "."
         << THRUST_MINOR_VERSION << "."
         << THRUST_SUBMINOR_VERSION << endl;
  }
  
  return HD_NO_ERROR;
}

hd_error hd_execute(hd_pipeline pl,
                    const hd_byte* h_filterbank, hd_size nsamps, hd_size nbits,
                    hd_size first_idx, hd_size nbeams, hd_size* nsamps_processed) {
  hd_error error = HD_NO_ERROR;
  
  Stopwatch total_timer;
  Stopwatch memory_timer;
  Stopwatch clean_timer;
  Stopwatch dedisp_timer;
  Stopwatch communicate_timer;
  Stopwatch copy_timer;
  Stopwatch baseline_timer;
  Stopwatch normalise_timer;
  Stopwatch filter_timer;
  Stopwatch coinc_timer;
  Stopwatch giants_timer;
  Stopwatch candidates_timer;
  
  start_timer(total_timer);
  
  start_timer(clean_timer);
  // Note: Filterbank cleaning must be done out-of-place
  hd_size nbytes = nsamps * pl->params.nchans * nbits / 8;
  start_timer(memory_timer);
  //pl->h_clean_filterbank.resize(nbytes);
  pl->h_clean_filterbank.resize(nbytes, 0);
  std::vector<int>          h_killmask(pl->params.nchans, 1);
  stop_timer(memory_timer);
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tCleaning 0-DM filterbank..." << endl;
  }
  
  // Start by cleaning up the filterbank based on the zero-DM time series
  hd_float cleaning_dm = 0.f;
  if( pl->params.verbosity >= 3 ) {
    /*
    cout << "\tWriting dirty filterbank to disk..." << endl;
    write_host_filterbank(&h_filterbank[0],
                          pl->params.nchans, nsamps, nbits,
                          pl->params.dt, pl->params.f0, pl->params.df,
                          "dirty_filterbank.fil");
    */
  }
  // Note: We only clean the narrowest zero-DM signals; otherwise we
  //         start removing real stuff from higher DMs.
  // Commenting out for now... VR
  /*error = clean_filterbank_rfi(pl->dedispersion_plan,
                               &h_filterbank[0],
                               nsamps,
                               nbits,
                               &pl->h_clean_filterbank[0],
                               &h_killmask[0],
                               cleaning_dm,
                               pl->params.dt,
                               pl->params.baseline_length,
                               pl->params.rfi_tol,
                               pl->params.rfi_min_beams,
                               1);//pl->params.boxcar_max);
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }*/
  std::copy(&h_filterbank[0],&h_filterbank[nsamps*pl->params.nchans],pl->h_clean_filterbank.begin());

  if( pl->params.verbosity >= 2 ) {
    cout << "Applying manual killmasks" << endl;
  }

  error = apply_manual_killmasks (pl->dedispersion_plan,
                                  &h_killmask[0], 
                                  pl->params.num_channel_zaps,
                                  pl->params.channel_zaps);
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }

  hd_size good_chan_count = thrust::reduce(h_killmask.begin(),
                                           h_killmask.end());
  hd_size bad_chan_count = pl->params.nchans - good_chan_count;
  if( pl->params.verbosity >= 2 ) {
    cout << "Bad channel count = " << bad_chan_count << endl;
  }

  // TESTING
  //h_clean_filterbank.assign(h_filterbank, h_filterbank+nbytes);
  
  stop_timer(clean_timer);
  
  if( pl->params.verbosity >= 3 ) {
    /*
    cout << "\tWriting killmask to disk..." << endl;
    std::ofstream killfile("killmask.dat");
    for( size_t i=0; i<h_killmask.size(); ++i ) {
      killfile << h_killmask[i] << "\n";
    }
    killfile.close();
    
    cout << "\tWriting cleaned filterbank to disk..." << endl;
    write_host_filterbank(&pl->h_clean_filterbank[0],
                          pl->params.nchans, nsamps, nbits,
                          pl->params.dt, pl->params.f0, pl->params.df,
                          "clean_filterbank.fil");
    */
  }
  if( pl->params.verbosity >= 2 ) {
    cout << "\tGenerating DM list..." << endl;
  }
  
  if( pl->params.verbosity >= 3 ) {
    cout << "dm_min = " << pl->params.dm_min << endl;
    cout << "dm_max = " << pl->params.dm_max << endl;
    cout << "dm_tol = " << pl->params.dm_tol << endl;
    cout << "dm_pulse_width = " << pl->params.dm_pulse_width << endl;
    cout << "nchans = " << pl->params.nchans << endl;
    cout << "dt = " << pl->params.dt << endl;
    
    cout << "dedisp nchans = " << dedisp_get_channel_count(pl->dedispersion_plan) << endl;
    cout << "dedisp dt = " << dedisp_get_dt(pl->dedispersion_plan) << endl;
    cout << "dedisp f0 = " << dedisp_get_f0(pl->dedispersion_plan) << endl;
    cout << "dedisp df = " << dedisp_get_df(pl->dedispersion_plan) << endl;
  }
  
  hd_size      dm_count = dedisp_get_dm_count(pl->dedispersion_plan);
  const float* dm_list  = dedisp_get_dm_list(pl->dedispersion_plan);
  
  const dedisp_size* scrunch_factors =
    dedisp_get_dt_factors(pl->dedispersion_plan);
  if (pl->params.verbosity >= 3 ) 
  {
    cout << "DM List for " << pl->params.dm_min << " to " << pl->params.dm_max << endl;
    for( hd_size i=0; i<dm_count; ++i ) {
      cout << dm_list[i] << endl;
    }
  }  

  if( pl->params.verbosity >= 2 ) {
    cout << "Scrunch factors:" << endl;
    for( hd_size i=0; i<dm_count; ++i ) {
      cout << scrunch_factors[i] << " ";
    }
    cout << endl;
  }
  
  // Set channel killmask for dedispersion
  dedisp_set_killmask(pl->dedispersion_plan, &h_killmask[0]);
  
  hd_size nsamps_computed  = nsamps - dedisp_get_max_delay(pl->dedispersion_plan);
  hd_size series_stride    = nsamps_computed;
  
  // Report the number of samples that will be properly processed
  *nsamps_processed = nsamps - (nbeams * (pl->params.boxcar_max + dedisp_get_max_delay(pl->dedispersion_plan)));
  if (nsamps < nbeams * (pl->params.boxcar_max + dedisp_get_max_delay(pl->dedispersion_plan)))
     *nsamps_processed = 0;

  //*nsamps_processed = nsamps_computed - (nbeams*pl->params.boxcar_max + (nbeams-1)*dedisp_get_max_delay(pl->dedispersion_plan));
  
  if( pl->params.verbosity >= 3 )
  {
    cout << "dm_count = " << dm_count << endl;
    cout << "max delay = " << dedisp_get_max_delay(pl->dedispersion_plan) << endl;
    cout << "nsamps_computed = " << nsamps_computed << endl;
    cout << "nsamps_processed = " << *nsamps_processed << endl;
  }
  
  hd_size beam = pl->params.beam;
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tAllocating memory for pipeline computations..." << endl;
  }
  
  start_timer(memory_timer);
  
  if( pl->params.verbosity >= 2 )
  {
    cerr << "series_stride == nsamps_computed = " << series_stride << " dm_count=" << dm_count << endl;
    cerr << "pl->h_dm_series.resize(" << series_stride * pl->params.dm_nbits/8 * dm_count << ")" << endl;
  }
  pl->h_dm_series.resize(series_stride * pl->params.dm_nbits/8 * dm_count, 0);
  pl->d_time_series.resize(series_stride, 0);
  pl->d_filtered_series.resize(series_stride, 0);
  
  stop_timer(memory_timer);
  
  RemoveBaselinePlan          baseline_remover;
  GetRMSPlan                  rms_getter;
  MatchedFilterPlan<hd_float> matched_filter_plan;
  GiantFinder                 giant_finder;
  
  thrust::device_vector<hd_float> d_giant_peaks;
  thrust::device_vector<hd_size>  d_giant_inds;
  thrust::device_vector<hd_size>  d_giant_begins;
  thrust::device_vector<hd_size>  d_giant_ends;
  thrust::device_vector<hd_size>  d_giant_filter_inds;
  thrust::device_vector<hd_size>  d_giant_dm_inds;
  thrust::device_vector<hd_size>  d_giant_members;
  
  typedef thrust::device_ptr<hd_float> dev_float_ptr;
  typedef thrust::device_ptr<hd_size>  dev_size_ptr;
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tDedispersing for DMs " << dm_list[0]
         << " to " << dm_list[dm_count-1] << "..." << endl;
  }
  
  // Dedisperse
  dedisp_error       derror;
  const dedisp_byte* in = &pl->h_clean_filterbank[0];
  dedisp_byte*       out = &pl->h_dm_series[0];
  dedisp_size        in_nbits = nbits;
  dedisp_size        in_stride = pl->params.nchans * in_nbits/8;
  dedisp_size        out_nbits = pl->params.dm_nbits;
  dedisp_size        out_stride = series_stride * out_nbits/8;
  unsigned           flags = 0;
  start_timer(dedisp_timer);
  derror = dedisp_execute_adv(pl->dedispersion_plan, nsamps,
                              in, in_nbits, in_stride,
                              out, out_nbits, out_stride,
                              flags);
  stop_timer(dedisp_timer);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  
  if( beam == 0 && first_idx == 0 ) {
    // TESTING
    //write_host_time_series((unsigned int*)out, nsamps_computed, out_nbits,
    //                       pl->params.dt, "dedispersed_0.tim");
  }
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tBeginning inner pipeline..." << endl;
  }
  
  // TESTING
  hd_size write_dm = 0;
  
  bool too_many_giants = false;
  
  // For each DM
  for( hd_size dm_idx=0; dm_idx<dm_count; ++dm_idx ) {
    hd_size  cur_dm_scrunch = scrunch_factors[dm_idx];
    hd_size  cur_nsamps  = nsamps_computed / cur_dm_scrunch;
    hd_float cur_dt      = pl->params.dt * cur_dm_scrunch;
    
    // Bail if the candidate rate is too high
    if( too_many_giants ) {
      break;
    }
    
    if( pl->params.verbosity >= 4 ) {
      cout << "dm_idx     = " << dm_idx << endl;
      cout << "scrunch    = " << scrunch_factors[dm_idx] << endl;
      cout << "cur_nsamps = " << cur_nsamps << endl;
      cout << "dt0        = " << pl->params.dt << endl;
      cout << "cur_dt     = " << cur_dt << endl;
        
      cout << "\tBaselining and normalising each beam..." << endl;
    }
    
    hd_float* time_series = thrust::raw_pointer_cast(&pl->d_time_series[0]);
    
    // Copy the time series to the device and convert to floats
    hd_size offset = dm_idx * series_stride * pl->params.dm_nbits/8;
    start_timer(copy_timer);
    switch( pl->params.dm_nbits ) {
    case 8:
      thrust::copy((unsigned char*)&pl->h_dm_series[offset],
                   (unsigned char*)&pl->h_dm_series[offset] + cur_nsamps,
                   pl->d_time_series.begin());
      break;
    case 16:
      thrust::copy((unsigned short*)&pl->h_dm_series[offset],
                   (unsigned short*)&pl->h_dm_series[offset] + cur_nsamps,
                   pl->d_time_series.begin());
      break;
    case 32:
      // Note: 32-bit implies float, not unsigned int
      //cerr  << "copying from h_dm_series to d_time_series [" << (&pl->h_dm_series[offset] + cur_nsamps) - (&pl->h_dm_series[offset]) << "]" << endl;
      thrust::copy( (float*)&pl->h_dm_series[offset],
                   ((float*)&pl->h_dm_series[offset]) + cur_nsamps,
                    pl->d_time_series.begin());
      break;
    default:
      return HD_INVALID_NBITS;
    }
    stop_timer(copy_timer);
    
    // Remove the baseline
    // -------------------
    // Note: Divided by 2 to form a smoothing radius
    hd_size nsamps_smooth = hd_size(pl->params.baseline_length /
                                    (2 * cur_dt));
    // Crop the smoothing length in case not enough samples
    start_timer(baseline_timer);
    
    // TESTING
    error = baseline_remover.exec(time_series, cur_nsamps, nsamps_smooth);
    stop_timer(baseline_timer);
    if( error != HD_NO_ERROR ) {
      return throw_error(error);
    }
    
    if( beam == 0 && dm_idx == write_dm && first_idx == 0 ) {
      // TESTING
      //write_device_time_series(time_series, cur_nsamps,
      //                         cur_dt, "baselined.tim");
    }
    // -------------------
    
    // Normalise
    // ---------
    start_timer(normalise_timer);
    hd_float rms = rms_getter.exec(time_series, cur_nsamps);
    thrust::transform(pl->d_time_series.begin(), pl->d_time_series.end(),
                      thrust::make_constant_iterator(hd_float(1.0)/rms),
                      pl->d_time_series.begin(),
                      thrust::multiplies<hd_float>());
    stop_timer(normalise_timer);
    
    if( beam == 0 && dm_idx == write_dm && first_idx == 0 ) {
      // TESTING
      //write_device_time_series(time_series, cur_nsamps,
      //                         cur_dt, "normalised.tim");
    }
    // ---------
    
    // Prepare the boxcar filters
    // --------------------------
    // We can't process the first and last max-filter-width/2 samples
    hd_size rel_boxcar_max = pl->params.boxcar_max/cur_dm_scrunch;
    
    hd_size max_nsamps_filtered = cur_nsamps + 1 - rel_boxcar_max;
    // This is the relative offset into the time series of the filtered data
    hd_size cur_filtered_offset = rel_boxcar_max / 2;
    
    // Create and prepare matched filtering operations
    start_timer(filter_timer);
    // Note: Filter width is relative to the current time resolution
    matched_filter_plan.prep(time_series, cur_nsamps, rel_boxcar_max);
    stop_timer(filter_timer);
    // --------------------------
    
    hd_float* filtered_series = thrust::raw_pointer_cast(&pl->d_filtered_series[0]);
    
    // Note: Filtering is done using a combination of tscrunching and
    //         'proper' boxcar convolution. The parameter min_tscrunch_width
    //         indicates how much of each to do. Raising min_tscrunch_width
    //         increases sensitivity but decreases performance and vice
    //         versa.
    
    // For each boxcar filter
    // Note: We cannot detect pulse widths < current time resolution
    for( hd_size filter_width=cur_dm_scrunch;
         filter_width<=pl->params.boxcar_max;
         filter_width*=2 ) {
      hd_size rel_filter_width = filter_width / cur_dm_scrunch;
      hd_size filter_idx = get_filter_index(filter_width);

//#define AJCHANGE
#ifdef AJCHANGE
      max_nsamps_filtered = cur_nsamps + 1 - filter_width;
      cur_filtered_offset = filter_width / 2;
#endif
      
      if( pl->params.verbosity >= 4 ) {
        cout << "Filtering each beam at width of " << filter_width << endl;
      }
      
      // Note: Filter width is relative to the current time resolution
      hd_size rel_min_tscrunch_width = std::max(pl->params.min_tscrunch_width
                                                / cur_dm_scrunch,
                                                hd_size(1));
      hd_size rel_tscrunch_width = std::max(2 * rel_filter_width
                                            / rel_min_tscrunch_width,
                                            hd_size(1));
      // Filter width relative to cur_dm_scrunch AND tscrunch
      hd_size rel_rel_filter_width = rel_filter_width / rel_tscrunch_width;
      
      start_timer(filter_timer);
      
      error = matched_filter_plan.exec(filtered_series,
                                       rel_filter_width,
                                       rel_tscrunch_width);
      
      if( error != HD_NO_ERROR ) {
        return throw_error(error);
      }
      // Divide and round up
      hd_size cur_nsamps_filtered = ((max_nsamps_filtered-1)
                                     / rel_tscrunch_width + 1);
      hd_size cur_scrunch = cur_dm_scrunch * rel_tscrunch_width;
      
      // Normalise the filtered time series (RMS ~ sqrt(time))
      // TODO: Avoid/hide the ugly thrust code?
      //         Consider making it a method of MatchedFilterPlan
      /*
      thrust::constant_iterator<hd_float> 
        norm_val_iter(1.0 / sqrt((hd_float)rel_filter_width));
      thrust::transform(thrust::device_ptr<hd_float>(filtered_series),
                        thrust::device_ptr<hd_float>(filtered_series)
                        + cur_nsamps_filtered,
                        norm_val_iter,
                        thrust::device_ptr<hd_float>(filtered_series),
                        thrust::multiplies<hd_float>());
      */
      // TESTING Proper normalisation
      hd_float rms = rms_getter.exec(filtered_series, cur_nsamps_filtered);
      thrust::transform(thrust::device_ptr<hd_float>(filtered_series),
                        thrust::device_ptr<hd_float>(filtered_series)
                        + cur_nsamps_filtered,
                        thrust::make_constant_iterator(hd_float(1.0)/rms),
                        thrust::device_ptr<hd_float>(filtered_series),
                        thrust::multiplies<hd_float>());

      stop_timer(filter_timer);
      
      if( beam == 0 && dm_idx == write_dm && first_idx == 0 &&
          filter_width == 8 ) {
        // TESTING
        //write_device_time_series(filtered_series,
        //                         cur_nsamps_filtered,
        //                         cur_dt, "filtered.tim");
      }
      
      hd_size prev_giant_count = d_giant_peaks.size();
      
      if( pl->params.verbosity >= 4 ) {
        cout << "Finding giants..." << endl;
      }
      
      start_timer(giants_timer);
      
      error = giant_finder.exec(filtered_series, cur_nsamps_filtered,
                                pl->params.detect_thresh,
                                //pl->params.cand_sep_time,
                                // Note: This was MB's recommendation
                                pl->params.cand_sep_time * rel_rel_filter_width,
                                d_giant_peaks,
                                d_giant_inds,
                                d_giant_begins,
                                d_giant_ends);
      
      if( error != HD_NO_ERROR ) {
        return throw_error(error);
      }
      
      hd_size rel_cur_filtered_offset = (cur_filtered_offset /
                                         rel_tscrunch_width);
      
      using namespace thrust::placeholders;
      thrust::transform(d_giant_inds.begin()+prev_giant_count,
                        d_giant_inds.end(),
                        d_giant_inds.begin()+prev_giant_count,
                        /*first_idx +*/ (_1+rel_cur_filtered_offset)*cur_scrunch);
      thrust::transform(d_giant_begins.begin()+prev_giant_count,
                        d_giant_begins.end(),
                        d_giant_begins.begin()+prev_giant_count,
                        /*first_idx +*/ (_1+rel_cur_filtered_offset)*cur_scrunch);
      thrust::transform(d_giant_ends.begin()+prev_giant_count,
                        d_giant_ends.end(),
                        d_giant_ends.begin()+prev_giant_count,
                        /*first_idx +*/ (_1+rel_cur_filtered_offset)*cur_scrunch);
      
      d_giant_filter_inds.resize(d_giant_peaks.size(), filter_idx);
      d_giant_dm_inds.resize(d_giant_peaks.size(), dm_idx);
      // Note: This could be used to track total member samples if desired
      d_giant_members.resize(d_giant_peaks.size(), 1);
      
      stop_timer(giants_timer);
      
      // Bail if the candidate rate is too high
      hd_size total_giant_count = d_giant_peaks.size();
      hd_float data_length_mins = nsamps * pl->params.dt / 60.0;
      if ( pl->params.max_giant_rate && ( total_giant_count / data_length_mins > pl->params.max_giant_rate ) ) {
        too_many_giants = true;
        float searched = ((float) dm_idx * 100) / (float) dm_count;
        cout << "WARNING: exceeded max giants/min, DM [" << dm_list[dm_idx] << "] space searched " << searched << "%" << endl;
        break;
      }
      
    } // End of filter width loop
  } // End of DM loop

  hd_size giant_count = d_giant_peaks.size();
  if( pl->params.verbosity >= 2 ) {
    cout << "Giant count = " << giant_count << endl;
  }
  
  start_timer(candidates_timer);

  thrust::host_vector<hd_float> h_group_peaks;
  thrust::host_vector<hd_size>  h_group_inds;
  thrust::host_vector<hd_size>  h_group_begins;
  thrust::host_vector<hd_size>  h_group_ends;
  thrust::host_vector<hd_size>  h_group_filter_inds;
  thrust::host_vector<hd_size>  h_group_dm_inds;
  thrust::host_vector<hd_size>  h_group_members;
  thrust::host_vector<hd_float> h_group_dms;

  if (!too_many_giants)
  {
    thrust::device_vector<hd_size> d_giant_labels(giant_count);
    hd_size* d_giant_labels_ptr = thrust::raw_pointer_cast(&d_giant_labels[0]);
  
    RawCandidates d_giants;
    d_giants.peaks = thrust::raw_pointer_cast(&d_giant_peaks[0]);
    d_giants.inds = thrust::raw_pointer_cast(&d_giant_inds[0]);
    d_giants.begins = thrust::raw_pointer_cast(&d_giant_begins[0]);
    d_giants.ends = thrust::raw_pointer_cast(&d_giant_ends[0]);
    d_giants.filter_inds = thrust::raw_pointer_cast(&d_giant_filter_inds[0]);
    d_giants.dm_inds = thrust::raw_pointer_cast(&d_giant_dm_inds[0]);
    d_giants.members = thrust::raw_pointer_cast(&d_giant_members[0]);
  
    hd_size filter_count = get_filter_index(pl->params.boxcar_max) + 1;

    if( pl->params.verbosity >= 2 ) {
      cout << "Grouping coincident candidates..." << endl;
    }
  
    hd_size label_count;
    hd_size nsamps_beam  = nsamps / nbeams;
    error = label_candidate_clusters(giant_count,
                                     *(ConstRawCandidates*)&d_giants,
                                     pl->params.cand_sep_time,
                                     pl->params.cand_sep_filter,
                                     pl->params.cand_sep_dm,
                                     nsamps_beam,
                                     d_giant_labels_ptr,
                                     &label_count);
    if( error != HD_NO_ERROR ) {
      return throw_error(error);
    }
  
    hd_size group_count = label_count;
    if( pl->params.verbosity >= 2 ) {
      cout << "Candidate count = " << group_count << endl;
    }
  
    thrust::device_vector<hd_float> d_group_peaks(group_count);
    thrust::device_vector<hd_size>  d_group_inds(group_count);
    thrust::device_vector<hd_size>  d_group_begins(group_count);
    thrust::device_vector<hd_size>  d_group_ends(group_count);
    thrust::device_vector<hd_size>  d_group_filter_inds(group_count);
    thrust::device_vector<hd_size>  d_group_dm_inds(group_count);
    thrust::device_vector<hd_size>  d_group_members(group_count);
  
    thrust::device_vector<hd_float> d_group_dms(group_count);
  
    RawCandidates d_groups;
    d_groups.peaks = thrust::raw_pointer_cast(&d_group_peaks[0]);
    d_groups.inds = thrust::raw_pointer_cast(&d_group_inds[0]);
    d_groups.begins = thrust::raw_pointer_cast(&d_group_begins[0]);
    d_groups.ends = thrust::raw_pointer_cast(&d_group_ends[0]);
    d_groups.filter_inds = thrust::raw_pointer_cast(&d_group_filter_inds[0]);
    d_groups.dm_inds = thrust::raw_pointer_cast(&d_group_dm_inds[0]);
    d_groups.members = thrust::raw_pointer_cast(&d_group_members[0]);
  
    merge_candidates(giant_count,
                     d_giant_labels_ptr,
                     *(ConstRawCandidates*)&d_giants,
                     d_groups);
  
    // Look up the actual DM of each group
    thrust::device_vector<hd_float> d_dm_list(dm_list, dm_list+dm_count);
    thrust::gather(d_group_dm_inds.begin(), d_group_dm_inds.end(),
                   d_dm_list.begin(),
                   d_group_dms.begin());
  
    // Device to host transfer of candidates
    h_group_peaks = d_group_peaks;
    h_group_inds = d_group_inds;
    h_group_begins = d_group_begins;
    h_group_ends = d_group_ends;
    h_group_filter_inds = d_group_filter_inds;
    h_group_dm_inds = d_group_dm_inds;
    h_group_members = d_group_members;
    h_group_dms = d_group_dms;
    //h_group_flags = d_group_flags;
  }
  
  if( pl->params.verbosity >= 2 ) {
    cout << "Writing output candidates, utc_start=" << pl->params.utc_start << endl;
  }

  char buffer[64];
  time_t now = pl->params.utc_start + (time_t) (first_idx / pl->params.spectra_per_second);
  strftime (buffer, 64, HD_TIMESTR, (struct tm*) gmtime(&now));

  std::stringstream ss;
  ss << std::setw(2) << std::setfill('0') << (pl->params.beam)+1;

  std::ostringstream oss;

  // number of samples between in each beam
  unsigned nsamps_beam  = nsamps / nbeams;
  // the number of samples that are valid (i.e. the max boxcar does not leak into the next beam)
  unsigned nsamps_valid = *nsamps_processed / nbeams;

  if ( pl->params.coincidencer_host != NULL && pl->params.coincidencer_port != -1 )
  {
    try 
    {

      unsigned n_events = 0;
      // count the number of valid events
      for (hd_size i=0; i<h_group_peaks.size(); ++i )
      {
        // for this candidate we must determine which beam it is in
        hd_size beam_idx = (hd_size) floor ((float) (h_group_inds[i]) / (float) nsamps_beam);

        hd_size beam_start = beam_idx * nsamps_beam + (pl->params.boxcar_max/2);
        hd_size beam_end   = beam_start + nsamps_valid;

        // if the event begins and ends within this beams data
        if ((beam_start <= h_group_inds[i]) && (h_group_inds[i] < beam_end))
          n_events++;
      }

      ClientSocket client_socket ( pl->params.coincidencer_host, pl->params.coincidencer_port );

      // send the UTC_START first
      strftime (buffer, 64, HD_TIMESTR, (struct tm*) gmtime(&(pl->params.utc_start)));
      oss <<  buffer << " ";

      // send the UTC timestamp of the first sample in this block
      time_t now = pl->params.utc_start + (time_t) (first_idx / pl->params.spectra_per_second);
      strftime (buffer, 64, HD_TIMESTR, (struct tm*) gmtime(&now));
      oss << buffer << " ";

      // send the "first_sample nbeams ncands"
      oss << first_idx << " ";
      oss << nbeams << " ";
      oss << n_events << endl;
      client_socket << oss.str();
      oss.flush();

      // reset oss
      oss.str("");

      for (hd_size i=0; i<h_group_peaks.size(); ++i ) 
      {
        // for this candidate we must determine which beam it is in
        hd_size beam_idx = (hd_size) floor ((float) (h_group_inds[i]) / (float) nsamps_beam);

        hd_size beam_start = beam_idx * nsamps_beam + (pl->params.boxcar_max/2);
        hd_size beam_end   = beam_start + nsamps_valid;

        // if the event begins and ends within this beams data
        if ((beam_start <= h_group_inds[i]) && (h_group_inds[i] < beam_end))
        {
          hd_size samp_offset = h_group_inds[i] - (nsamps_beam * beam_idx);
          hd_size samp_idx = first_idx + samp_offset;

          oss << h_group_peaks[i] << "\t"
              << samp_idx << "\t"
              << samp_idx * pl->params.dt << "\t"
              << h_group_filter_inds[i] << "\t"
              << h_group_dm_inds[i] << "\t"
              << h_group_dms[i] << "\t"
              << h_group_members[i] << "\t"
              << first_idx + h_group_begins[i] << "\t"
              << first_idx + h_group_ends[i] << "\t"
              << (pl->params.beam + beam_idx + 1) << endl;

          client_socket << oss.str();
          oss.flush();
          oss.str("");
        }
      }
      // client_socket should close when it goes out of scope...
    }
    catch (SocketException& e )
    {
      std::cerr << "SocketException was caught:" << e.description() << "\n";
    }
  }
  else
  {
    if( pl->params.verbosity >= 2 )
      cout << "Output timestamp: " << buffer << endl;

    std::string filename = std::string(pl->params.output_dir) + "/" + std::string(buffer) + "_" + ss.str() + ".cand";

    if( pl->params.verbosity >= 2 )
      cout << "Output filename: " << filename << endl;

    std::ofstream cand_file(filename.c_str(), std::ios::out);
    if( pl->params.verbosity >= 2 )
      cout << "Dumping " << h_group_peaks.size() << " candidates to " << filename << endl;

    if (cand_file.good())
    {
      for( hd_size i=0; i<h_group_peaks.size(); ++i ) 
      {
        // for this candidate we must determine which beam it is in
        hd_size beam_idx = (hd_size) floor ((float) (h_group_inds[i]) / (float) nsamps_beam);

        hd_size beam_start = beam_idx * nsamps_beam + (pl->params.boxcar_max/2);
        hd_size beam_end   = beam_start + nsamps_valid;

        // if the event begins and ends within this beams data
        if ((beam_start <= h_group_inds[i]) && (h_group_inds[i] < beam_end))
        {
          hd_size samp_offset = h_group_inds[i] - (nsamps_beam * beam_idx);
          hd_size samp_idx = first_idx + samp_offset;

          cand_file << h_group_peaks[i] << "\t"
                    << samp_idx << "\t"
                    << pl->params.dt * samp_idx << "\t"
                    << h_group_filter_inds[i] << "\t"
                    << h_group_dm_inds[i] << "\t"
                    << h_group_dms[i] << "\t"
                    << h_group_members[i] << "\t"
                    << first_idx + (h_group_begins[i] % nsamps_beam) << "\t"
                    << first_idx + (h_group_ends[i] % nsamps_beam)  << "\t"
                    << (pl->params.beam + beam_idx + 1) << "\t"
                    << "\n";
        }
      }
    }
    else
      cout << "Skipping dump due to bad file open on " << filename << endl;
    cand_file.close();
  }
    
  stop_timer(candidates_timer);
  
  stop_timer(total_timer);
  
#ifdef HD_BENCHMARK
  if( pl->params.verbosity >= 1 )
  {
  cout << "Mem alloc time:          " << memory_timer.getTime() << endl;
  cout << "0-DM cleaning time:      " << clean_timer.getTime() << endl;
  cout << "Dedispersion time:       " << dedisp_timer.getTime() << endl;
  cout << "Copy time:               " << copy_timer.getTime() << endl;
  cout << "Baselining time:         " << baseline_timer.getTime() << endl;
  cout << "Normalisation time:      " << normalise_timer.getTime() << endl;
  cout << "Filtering time:          " << filter_timer.getTime() << endl;
  cout << "Find giants time:        " << giants_timer.getTime() << endl;
  cout << "Process candidates time: " << candidates_timer.getTime() << endl;
  cout << "Total time:              " << total_timer.getTime() << endl;
  }

  hd_float time_sum = (memory_timer.getTime() +
                       clean_timer.getTime() +
                       dedisp_timer.getTime() +
                       copy_timer.getTime() +
                       baseline_timer.getTime() +
                       normalise_timer.getTime() +
                       filter_timer.getTime() +
                       giants_timer.getTime() +
                       candidates_timer.getTime());
  hd_float misc_time = total_timer.getTime() - time_sum;
  
  /*
  std::ofstream timing_file("timing.dat", std::ios::app);
  timing_file << total_timer.getTime() << "\t"
              << misc_time << "\t"
              << memory_timer.getTime() << "\t"
              << clean_timer.getTime() << "\t"
              << dedisp_timer.getTime() << "\t"
              << copy_timer.getTime() << "\t"
              << baseline_timer.getTime() << "\t"
              << normalise_timer.getTime() << "\t"
              << filter_timer.getTime() << "\t"
              << giants_timer.getTime() << "\t"
              << candidates_timer.getTime() << endl;
  timing_file.close();
  */
  
#endif // HD_BENCHMARK
  
  if( too_many_giants ) {
    return HD_TOO_MANY_EVENTS;
  }
  else {
    return HD_NO_ERROR;
  }
}

void hd_destroy_pipeline(hd_pipeline pipeline) {
  if( pipeline->params.verbosity >= 2 ) {
    cout << "\tDeleting pipeline object..." << endl;
  }
  
  dedisp_destroy_plan(pipeline->dedispersion_plan);
  
  // Note: This assumes memory owned by pipeline cleans itself up
  if( pipeline ) {
    delete pipeline;
  }
}
