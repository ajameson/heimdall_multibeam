/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "hd/find_giants.h"
#include "hd/cached_allocator.cuh"

// TESTING only
#include "hd/stopwatch.h"
#include <iostream>
//#define PRINT_BENCHMARKS

#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/adjacent_difference.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <thrust/iterator/retag.h>

// Global instance of the custom temporary memory allocator for Thrust
// TODO: We should be calling g_allocator.free_all() somewhere at the end of
//         the application to ensure memory is freed before the underlying
//         device backend (e.g., CUDART) goes out of scope. Not sure exactly
//         where to put it though.
cached_allocator g_allocator;

template<typename T>
struct greater_than_val : public thrust::unary_function<T, bool> {
  T val;
  greater_than_val(T val_) : val(val_) {}
  inline __host__ __device__
  bool operator()(T x) const {
    return x > val;
  }
};


typedef thrust::tuple<hd_float, hd_size> tplfloatint;
struct greater_than_val_mb : public thrust::unary_function<tplfloatint, bool> {
  hd_float  val;
  const hd_size   stride;
  const hd_size   count;
  greater_than_val_mb(hd_float val_, hd_size stride_, hd_size count_) 
    : val(val_), stride(stride_), count(count_) {}
  inline __host__ __device__
  bool operator()(const tplfloatint& x) const {
    hd_size i = thrust::get<1>(x);
    hd_size samp = i % stride;
    hd_float samp_val = thrust::get<0>(x);
    return (samp < count && samp_val > val);
  }
};

template<typename T>
struct maximum_first : public thrust::binary_function<T,T,T> {
  inline __host__ __device__
  T operator()(T a, T b) const {
    return thrust::get<0>(a) >= thrust::get<0>(b) ? a : b;
  }
};

template<typename T>
struct nearby : public thrust::binary_function<T,T,bool> {
  T max_dist;
  nearby(T max_dist_) : max_dist(max_dist_) {}
  inline __host__ __device__
  bool operator()(T a, T b) const {
    return b <= a + max_dist;
  }
};
template<typename T>
struct not_nearby : public thrust::binary_function<T,T,bool> {
  T max_dist;
  not_nearby(T max_dist_) : max_dist(max_dist_) {}
  inline __host__ __device__
  bool operator()(T b, T a) const {
    return b > a + max_dist;
  }
};

template<typename T>
struct plus_one : public thrust::unary_function<T,T> {
  inline __host__ __device__
  T operator()(T x) const { return x+1; }
};

class GiantFinder_impl {
  thrust::device_vector<hd_float> d_giant_data;
  thrust::device_vector<hd_size>  d_giant_data_inds;
  thrust::device_vector<int>      d_giant_data_segments;
  thrust::device_vector<hd_size>  d_giant_data_seg_ids;
public:
  hd_error exec(const hd_float* d_data, hd_size count, hd_size nbeams,
                hd_float thresh, hd_size merge_dist,
                hd_size beam_stride, hd_size beam_count,
                thrust::device_vector<hd_float>& d_giant_peaks,
                thrust::device_vector<hd_size>&  d_giant_inds,
                thrust::device_vector<hd_size>&  d_giant_begins,
                thrust::device_vector<hd_size>&  d_giant_ends) {

    // This algorithm works by extracting all samples in the time series
    //   above thresh (the giant_data), segmenting those samples into
    //   isolated giants (based on merge_dist), and then computing the
    //   details of each giant into the d_giant_* arrays using
    //   reduce_by_key and some scatter operations.
    
    using thrust::copy_if;
    using thrust::make_zip_iterator;
    using thrust::make_tuple;
    using thrust::make_counting_iterator;
  
    typedef thrust::device_ptr<const hd_float> const_float_ptr;
    typedef thrust::device_ptr<hd_float>             float_ptr;
    typedef thrust::device_ptr<hd_size>              size_ptr;
 
    const_float_ptr d_data_begin(d_data);
    const_float_ptr d_data_end(d_data + count);
  
#ifdef PRINT_BENCHMARKS
    Stopwatch timer;
  
    timer.start();
#endif
 
    // Note: The calls to Thrust in this function are retagged to use a
    //         custom temporary memory allocator (cached_allocator.cuh).
    //       This turns out to be critical to performance!
    // Quickly count how much giant data there is so we know the space needed
    hd_size giant_data_count = 
        thrust::count_if(
            make_zip_iterator(make_tuple(thrust::retag<my_tag>(d_data_begin), make_counting_iterator(0u))),
            make_zip_iterator(make_tuple(thrust::retag<my_tag>(d_data_begin), make_counting_iterator(0u)))+count,
            greater_than_val_mb(thresh, beam_stride, beam_count));

    // We can bail early if there are no giants at all
    if( 0 == giant_data_count ) {
      //std::cout << "**** Found ZERO giants" << std::endl;
      return HD_NO_ERROR;
    }
  
#ifdef PRINT_BENCHMARKS
    hipDeviceSynchronize();
    timer.stop();
    std::cout << "count_if time:           " << timer.getTime() << " s" << std::endl;
    timer.reset();
  
    timer.start();
#endif
  
    d_giant_data.resize(giant_data_count);
    d_giant_data_inds.resize(giant_data_count);
  
#ifdef PRINT_BENCHMARKS
    hipDeviceSynchronize();
    timer.stop();
    std::cout << "giant_data resize time:  " << timer.getTime() << " s" << std::endl;
    timer.reset();
  
    // Copy all of the giant data and their locations into one place
  
    timer.start();
#endif
  
    hd_size giant_data_count2 = 
      copy_if(make_zip_iterator(make_tuple(thrust::retag<my_tag>(d_data_begin),
                                           make_counting_iterator(0u))),
              make_zip_iterator(make_tuple(thrust::retag<my_tag>(d_data_begin),
                                           make_counting_iterator(0u)))+count,
              make_zip_iterator(make_tuple(thrust::retag<my_tag>(d_data_begin),
                                           make_counting_iterator(0u))),
              make_zip_iterator(make_tuple(thrust::retag<my_tag>(d_giant_data.begin()),
                                           thrust::retag<my_tag>(d_giant_data_inds.begin()))),
              greater_than_val_mb(thresh, beam_stride, beam_count))
      - make_zip_iterator(make_tuple(thrust::retag<my_tag>(d_giant_data.begin()),
                                     thrust::retag<my_tag>(d_giant_data_inds.begin())));
  
#ifdef PRINT_BENCHMARKS
    hipDeviceSynchronize();
    timer.stop();
    std::cout << "giant_data copy_if time: " << timer.getTime() << " s" << std::endl;
    timer.reset();
  
    timer.start();
#endif
  
    // Create an array of head flags indicating candidate segments
    //thrust::device_vector<int> d_giant_data_segments(giant_data_count);
    d_giant_data_segments.resize(giant_data_count);
    thrust::adjacent_difference(thrust::retag<my_tag>(d_giant_data_inds.begin()),
                                thrust::retag<my_tag>(d_giant_data_inds.end()),
                                thrust::retag<my_tag>(d_giant_data_segments.begin()),
                                not_nearby<hd_size>(merge_dist));
  
    //hd_size giant_count_quick = thrust::count(d_giant_data_segments.begin(),
    //                                          d_giant_data_segments.end(),
    //                                          (int)true);
  
    // The first element is implicitly a segment head
    if( giant_data_count > 0 ) {
      d_giant_data_segments.front() = 0;
      //d_giant_data_segments.front() = 1;
    }
  
    //thrust::device_vector<hd_size> d_giant_data_seg_ids(d_giant_data_segments.size());
    d_giant_data_seg_ids.resize(d_giant_data_segments.size());
    
    thrust::inclusive_scan(thrust::retag<my_tag>(d_giant_data_segments.begin()),
                           thrust::retag<my_tag>(d_giant_data_segments.end()),
                           thrust::retag<my_tag>(d_giant_data_seg_ids.begin()));
  
    // We extract the number of giants from the end of the exclusive scan
    //hd_size giant_count = d_giant_data_seg_ids.back() +
    //  d_giant_data_segments.back() + 1;
    hd_size giant_count = d_giant_data_seg_ids.back() + 1;
    //hd_size giant_count = d_giant_data_seg_ids.back() +
    //  d_giant_data_segments.back();
  
    // Report back the actual number of giants found
    //total_giant_count = giant_count;
    
#ifdef PRINT_BENCHMARKS
    hipDeviceSynchronize();
    timer.stop();
    std::cout << "giant segments time:     " << timer.getTime() << " s" << std::endl;
    timer.reset();
  
    timer.start();
#endif
  
    hd_size new_giants_offset = d_giant_peaks.size();
    // Allocate space for the new giants
    d_giant_peaks.resize(d_giant_peaks.size() + giant_count);
    d_giant_inds.resize(d_giant_inds.size() + giant_count);
    d_giant_begins.resize(d_giant_begins.size() + giant_count);
    d_giant_ends.resize(d_giant_ends.size() + giant_count);
    float_ptr new_giant_peaks_begin(&d_giant_peaks[new_giants_offset]);
    size_ptr  new_giant_inds_begin(&d_giant_inds[new_giants_offset]);
    size_ptr  new_giant_begins_begin(&d_giant_begins[new_giants_offset]);
    size_ptr  new_giant_ends_begin(&d_giant_ends[new_giants_offset]);
  
#ifdef PRINT_BENCHMARKS
    hipDeviceSynchronize();
    timer.stop();
    std::cout << "giants resize time:      " << timer.getTime() << " s" << std::endl;
    timer.reset();
  
    timer.start();
#endif
    
  
    // Now we find the value (snr) and location (time) of each giant's maximum
    hd_size giant_count2 = 
      reduce_by_key(thrust::retag<my_tag>(d_giant_data_inds.begin()), // the keys
                    thrust::retag<my_tag>(d_giant_data_inds.end()),
                    make_zip_iterator(make_tuple(thrust::retag<my_tag>(d_giant_data.begin()),
                                                 thrust::retag<my_tag>(d_giant_data_inds.begin()))),
                    thrust::make_discard_iterator(), // the keys output
                    make_zip_iterator(make_tuple(thrust::retag<my_tag>(new_giant_peaks_begin),
                                                 thrust::retag<my_tag>(new_giant_inds_begin))),
                    nearby<hd_size>(merge_dist),
                    maximum_first<thrust::tuple<hd_float,hd_size> >())
      .second - make_zip_iterator(make_tuple(thrust::retag<my_tag>(new_giant_peaks_begin),
                                             thrust::retag<my_tag>(new_giant_inds_begin)));
  
#ifdef PRINT_BENCHMARKS
    hipDeviceSynchronize();
    timer.stop();
    std::cout << "reduce_by_key time:      " << timer.getTime() << " s" << std::endl;
    timer.reset();
  
    timer.start();
#endif
  
    // Now we make the first segment explicit
    if( giant_count > 0 ) {
      d_giant_data_segments[0] = 1;
    }
  
    // Create arrays of the beginning and end indices of each giant
    thrust::scatter_if(d_giant_data_inds.begin(),
                       d_giant_data_inds.end(),
                       d_giant_data_seg_ids.begin(),
                       d_giant_data_segments.begin(),
                       new_giant_begins_begin);
    thrust::scatter_if(make_transform_iterator(d_giant_data_inds.begin(),
                                                                     plus_one<hd_size>()),
                       make_transform_iterator(d_giant_data_inds.end() - 1,
                                                                     plus_one<hd_size>()),
                       d_giant_data_seg_ids.begin(),
                       d_giant_data_segments.begin() + 1,
                       new_giant_ends_begin);
    
    if( giant_count > 0 ) {
      d_giant_ends.back() = d_giant_data_inds.back() + 1;
    }
  
  
#ifdef PRINT_BENCHMARKS
    hipDeviceSynchronize();
    timer.stop();
    std::cout << "begin/end copy_if time:  " << timer.getTime() << " s" << std::endl;
    timer.reset();
  
    std::cout << "--------------------" << std::endl;
#endif
  
    return HD_NO_ERROR;
  }
  
};

// Public interface (wrapper for implementation)
GiantFinder::GiantFinder()
  : m_impl(new GiantFinder_impl) {}
hd_error GiantFinder::exec(const hd_float* d_data, hd_size count, hd_size nbeams,
                           hd_float thresh, hd_size merge_dist,
                           hd_size beam_stride, hd_size beam_count,
                           thrust::device_vector<hd_float>& d_giant_peaks,
                           thrust::device_vector<hd_size>&  d_giant_inds,
                           thrust::device_vector<hd_size>&  d_giant_begins,
                           thrust::device_vector<hd_size>&  d_giant_ends) {
  return m_impl->exec(d_data, count, nbeams,
                      thresh, merge_dist,
                      beam_stride, beam_count,
                      d_giant_peaks,
                      d_giant_inds,
                      d_giant_begins,
                      d_giant_ends);
}
